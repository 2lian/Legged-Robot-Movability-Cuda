#include "hip/hip_runtime.h"
#include "cross_compiled.cuh"
#include "unified_math_cuda.cu.h"
#include "one_leg.cu.h"
#define BLOCSIZE 1024 / 4

#define CUDA_CHECK_ERROR(errorMessage)                                                   \
    do {                                                                                 \
        hipError_t err = hipGetLastError();                                            \
        if (err != hipSuccess) {                                                        \
            fprintf(stderr, "CUDA error in %s: %s\n", errorMessage,                      \
                    hipGetErrorString(err));                                            \
            exit(EXIT_FAILURE);                                                          \
        }                                                                                \
    } while (0)

/**
 * @brief applies kernel on provided array to provided array
 *
 * @tparam T_in
 * @tparam T_out
 * @param input Input array of any type
 * @param dim dimension of the leg
 * @param kernel cuda kernel to be used
 * @param output result is stored here
 */
template <typename T_in, typename param, typename T_out>
float apply_kernel(const Array<T_in> input, const param dim,
                   void (*kernel)(const Array<T_in>, const param, Array<T_out> const),
                   Array<T_out> const output) {
    Array<T_in> gpu_in{};
    Array<T_out> gpu_out{};
    gpu_in.length = input.length;
    gpu_out.length = output.length;
    hipMalloc(&gpu_in.elements, gpu_in.length * sizeof(T_in));
    CUDA_CHECK_ERROR("hipMalloc gpu_in.elements");
    hipMalloc(&gpu_out.elements, gpu_out.length * sizeof(T_out));
    CUDA_CHECK_ERROR("hipMalloc gpu_out.elements");

    hipMemcpy(gpu_in.elements, input.elements, gpu_in.length * sizeof(T_in),
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR("hipMemcpy gpu_in.elements");

    constexpr int blockSize = BLOCSIZE;
    int numBlock = (input.length + blockSize - 1) / blockSize;
    // Prepare
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);
    // Do something on GPU
    kernel<<<numBlock, blockSize>>>(gpu_in, dim, gpu_out);
    // Stop event and sync
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_CHECK_ERROR("Kernel launch");

    hipMemcpy(output.elements, gpu_out.elements, output.length * sizeof(T_out),
               hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR("hipMemcpy gpu_out.elements");
    hipDeviceSynchronize();

    hipFree(gpu_in.elements);
    hipFree(gpu_out.elements);
    return elapsedTime;
}

template <typename T_in, typename param, typename T_out>
float apply_recurs(const Array<T_in> input, const param dim, Array<T_out> const output) {
    Array<T_in> gpu_in{};
    Array<T_out> gpu_out{};
    gpu_in.length = input.length;
    gpu_out.length = output.length;
    hipMalloc(&gpu_in.elements, gpu_in.length * sizeof(T_in));
    CUDA_CHECK_ERROR("hipMalloc gpu_in.elements");
    hipMalloc(&gpu_out.elements, gpu_out.length * sizeof(T_out));
    CUDA_CHECK_ERROR("hipMalloc gpu_out.elements");

    hipMemcpy(gpu_in.elements, input.elements, gpu_in.length * sizeof(T_in),
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR("hipMemcpy gpu_in.elements");

    constexpr int blockSize = BLOCSIZE;
    int numBlock = (input.length + blockSize - 1) / blockSize;
    Box box;
    box.center = make_float3(200, 0, 0);
    box.topOffset = make_float3(400, 1, 500);
    const uint max_quad_ind = pow(8, 3);
    numBlock = (max_quad_ind + blockSize - 1) / blockSize;
    // Prepare
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);
    // Do something on GPU
    recursive_kernel<<<numBlock, blockSize>>>(box, gpu_in, dim, gpu_out, 0);
    // Stop event and sync
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_CHECK_ERROR("Kernel launch");

    hipMemcpy(output.elements, gpu_out.elements, output.length * sizeof(T_out),
               hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR("hipMemcpy gpu_out.elements");
    hipDeviceSynchronize();

    hipFree(gpu_in.elements);
    hipFree(gpu_out.elements);
    return elapsedTime;
}
template float apply_recurs<float3, LegDimensions, float3>(Array<float3>,
                                                               LegDimensions,
                                                               Array<float3>);
// Explicit instantiation for float3, float3
template float apply_kernel<float3, LegDimensions, float3>(
    Array<float3>, LegDimensions, void (*)(Array<float3>, LegDimensions, Array<float3>),
    Array<float3>);

// Explicit instantiation for float3, bool
template float apply_kernel<float3, LegDimensions, bool>(
    Array<float3>, LegDimensions,
    void (*)(const Array<float3>, LegDimensions, Array<bool>), Array<bool>);

// Explicit instantiation for float3, float3
template float apply_kernel<float3, LegCompact, float3>(
    Array<float3>, LegCompact, void (*)(Array<float3>, LegCompact, Array<float3>),
    Array<float3>);

// Explicit instantiation for float3, bool
template float apply_kernel<float3, LegCompact, bool>(Array<float3>, LegCompact,
                                                      void (*)(const Array<float3>,
                                                               LegCompact, Array<bool>),
                                                      Array<bool>);
