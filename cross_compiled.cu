#include "hip/hip_runtime.h"
#include "cross_compiled.cuh"

#define CUDA_CHECK_ERROR(errorMessage)                                         \
    do {                                                                       \
        hipError_t err = hipGetLastError();                                  \
        if (err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error in %s: %s\n", errorMessage,            \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

/**
 * @brief applies kernel on provided array to provided array
 *
 * @tparam T_in
 * @tparam T_out
 * @param input Input array of any type
 * @param dim dimension of the leg
 * @param kernel cuda kernel to be used
 * @param output result is stored here
 */
template <typename T_in, typename param, typename T_out>
void apply_kernel(const Array<T_in> input, const param dim,
                  void (*kernel)(const Array<T_in>, const param,
                                 Array<T_out> const),
                  Array<T_out> const output) {
    Array<T_in> gpu_in{};
    Array<T_out> gpu_out{};
    gpu_in.length = input.length;
    gpu_out.length = output.length;
    hipMalloc(&gpu_in.elements, gpu_in.length * sizeof(T_in));
    CUDA_CHECK_ERROR("hipMalloc gpu_in.elements");
    hipMalloc(&gpu_out.elements, gpu_out.length * sizeof(T_out));
    CUDA_CHECK_ERROR("hipMalloc gpu_out.elements");

    hipMemcpy(gpu_in.elements, input.elements, gpu_in.length * sizeof(T_in),
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR("hipMemcpy gpu_in.elements");

    int blockSize = 1024;
    int numBlock = (input.length + blockSize - 1) / blockSize;
    kernel<<<numBlock, blockSize>>>(gpu_in, dim, gpu_out);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Kernel launch");

    hipMemcpy(output.elements, gpu_out.elements, output.length * sizeof(T_out),
               hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR("hipMemcpy gpu_out.elements");
    hipDeviceSynchronize();

    hipFree(gpu_in.elements);
    hipFree(gpu_out.elements);
}

// Explicit instantiation for float3, float3
template void apply_kernel<float3, LegDimensions, float3>(
    Array<float3>, LegDimensions,
    void (*)(Array<float3>, LegDimensions, Array<float3>), Array<float3>);

// Explicit instantiation for float3, bool
template void apply_kernel<float3, LegDimensions, bool>(Array<float3>, LegDimensions,
                                         void (*)(const Array<float3>,
                                                  LegDimensions, Array<bool>),
                                         Array<bool>);

// Explicit instantiation for float3, float3
template void apply_kernel<float3, LegCompact, float3>(
    Array<float3>, LegCompact,
    void (*)(Array<float3>, LegCompact, Array<float3>), Array<float3>);

// Explicit instantiation for float3, bool
template void apply_kernel<float3, LegCompact, bool>(Array<float3>, LegCompact,
                                         void (*)(const Array<float3>,
                                                  LegCompact, Array<bool>),
                                         Array<bool>);
