#include "hip/hip_runtime.h"
#include "cross_compiled.cuh"
#include "one_leg.cu.h"
#include "settings.h"
#include "unified_math_cuda.cu.h"
#include <chrono>
#include <iostream>
#include <ostream>
#include <thread>
#define BLOCSIZE 1024 / 4
using namespace std::chrono_literals;

#define CUDA_CHECK_ERROR(errorMessage)                                                   \
    do {                                                                                 \
        hipError_t err = hipGetLastError();                                            \
        if (err != hipSuccess) {                                                        \
            fprintf(stderr, "CUDA error in %s: %s\n", errorMessage,                      \
                    hipGetErrorString(err));                                            \
            exit(EXIT_FAILURE);                                                          \
        }                                                                                \
    } while (0)

/**
 * @brief applies kernel on provided array to provided array
 *
 * @tparam T_in
 * @tparam T_out
 * @param input Input array of any type
 * @param dim dimension of the leg
 * @param kernel cuda kernel to be used
 * @param output result is stored here
 */
template <typename T_in, typename param, typename T_out>
float apply_kernel(const Array<T_in> input, const param dim,
                   void (*kernel)(const Array<T_in>, const param, Array<T_out> const),
                   Array<T_out> const output) {
    Array<T_in> gpu_in{};
    Array<T_out> gpu_out{};
    gpu_in.length = input.length;
    gpu_out.length = output.length;
    hipMalloc(&gpu_in.elements, gpu_in.length * sizeof(T_in));
    CUDA_CHECK_ERROR("hipMalloc gpu_in.elements");
    hipMalloc(&gpu_out.elements, gpu_out.length * sizeof(T_out));
    CUDA_CHECK_ERROR("hipMalloc gpu_out.elements");

    hipMemcpy(gpu_in.elements, input.elements, gpu_in.length * sizeof(T_in),
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR("hipMemcpy gpu_in.elements");

    constexpr int blockSize = BLOCSIZE;
    int numBlock = (input.length + blockSize - 1) / blockSize;
    kernel<<<numBlock, blockSize>>>(gpu_in, dim, gpu_out); // warmup
    // Prepare
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);
    // Do something on GPU
    kernel<<<numBlock, blockSize>>>(gpu_in, dim, gpu_out);
    // Stop event and sync
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_CHECK_ERROR("Kernel launch");

    hipMemcpy(output.elements, gpu_out.elements, output.length * sizeof(T_out),
               hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR("hipMemcpy gpu_out.elements");
    hipDeviceSynchronize();

    hipFree(gpu_in.elements);
    hipFree(gpu_out.elements);
    return elapsedTime;
}

template <typename T_in, typename param, typename T_out>
float apply_recurs(const Array<T_in> input, const param dim, Array<T_out> const output) {
    Array<T_in> gpu_in{};
    Array<T_out> gpu_out{};
    gpu_in.length = input.length;
    gpu_out.length = output.length;
    hipMalloc(&gpu_in.elements, gpu_in.length * sizeof(T_in));
    CUDA_CHECK_ERROR("hipMalloc gpu_in.elements");
    hipMalloc(&gpu_out.elements, gpu_out.length * sizeof(T_out));
    CUDA_CHECK_ERROR("hipMalloc gpu_out.elements");

    hipMemcpy(gpu_in.elements, input.elements, gpu_in.length * sizeof(T_in),
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR("hipMemcpy gpu_in.elements");

    constexpr int blockSize = BLOCSIZE;
    int numBlock = (input.length + blockSize - 1) / blockSize;
    Box box;
    box.center = make_float3(BoxCenter[0], BoxCenter[1], BoxCenter[2]);
    box.topOffset = make_float3(BoxSize[0], BoxSize[1], BoxSize[2]);
    const uint max_quad_ind = pow(8, 2);
    numBlock = (max_quad_ind + blockSize - 1) / blockSize;
    // recursive_kernel<<<1, 24>>>(box, gpu_in, dim, gpu_out, 30);
    // Prepare
    hipEvent_t start, stop;
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, stream);
    // Do something on GPU
    recursive_kernel<<<numBlock, blockSize, 0, stream>>>(box, gpu_in, dim, gpu_out, 0, 0,
                                                         false);
    // recursive_kernel<<<numBlock, blockSize>>>(box, gpu_in, dim, gpu_out, 0, 0, false);
    // recursive_kernel<<<1, 24>>>(box, gpu_in, dim, gpu_out, 0, 0);
    // Stop event and sync
    // hipDeviceSynchronize();
    // std::this_thread::sleep_for(1s);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_CHECK_ERROR("Kernel launch");

    hipMemcpy(output.elements, gpu_out.elements, output.length * sizeof(T_out),
               hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR("hipMemcpy gpu_out.elements");
    hipDeviceSynchronize();

    hipFree(gpu_in.elements);
    hipFree(gpu_out.elements);
    return elapsedTime;
}
template float apply_recurs<float3, LegDimensions, float3>(Array<float3>, LegDimensions,
                                                           Array<float3>);
// Explicit instantiation for float3, float3
template float apply_kernel<float3, LegDimensions, float3>(
    Array<float3>, LegDimensions, void (*)(Array<float3>, LegDimensions, Array<float3>),
    Array<float3>);

// Explicit instantiation for float3, bool
template float apply_kernel<float3, LegDimensions, bool>(
    Array<float3>, LegDimensions,
    void (*)(const Array<float3>, LegDimensions, Array<bool>), Array<bool>);

// Explicit instantiation for float3, float3
template float apply_kernel<float3, LegCompact, float3>(
    Array<float3>, LegCompact, void (*)(Array<float3>, LegCompact, Array<float3>),
    Array<float3>);

// Explicit instantiation for float3, bool
template float apply_kernel<float3, LegCompact, bool>(Array<float3>, LegCompact,
                                                      void (*)(const Array<float3>,
                                                               LegCompact, Array<bool>),
                                                      Array<bool>);

__host__ double apply_dist_cpu(const Array<float3> input, const LegDimensions dim,
                           Array<float3> const output) {
    auto start = std::chrono::high_resolution_clock::now();
    distance_kernel_cpu(input, dim, output);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = (end - start)*1000;
    return duration.count();
}

__host__ double apply_reach_cpu(const Array<float3> input, const LegDimensions dim,
                           Array<bool> const output) {
    auto start = std::chrono::high_resolution_clock::now();
    // std::cout << "start" << std::endl;
    reachability_kernel_cpu(input, dim, output);
    // std::cout << "end" << std::endl;
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = (end - start)*1000;
    return duration.count();
}
