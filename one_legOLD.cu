#include "hip/hip_runtime.h"
#include "HeaderCPP.h"
#include "HeaderCUDA.h"
// #include "circles.cu.h"
// #include "one_leg.cu.h"

__device__ __forceinline__ void clamp_on_circle(const float* center, const float& radius,
                                                const bool& clamp_direction, float& x,
                                                float& y) {
    x -= center[0];
    y -= center[1];
    float magnitude = sqrtf(x * x + y * y);
    bool radius_direction = !signbit(radius - magnitude);

    if (clamp_direction != radius_direction or magnitude < 0.001f) {
        x = 0;
        y = 0;
        return;
    }

    x -= radius * x / magnitude;
    y -= radius * y / magnitude;
}

__device__ __forceinline__ void place_on_vert_plane(float& x, float& z,
                                                    const LegDimensions& dim) {
    // Femur as the frame of reference witout rotation
    x -= dim.coxa_length;

    float femur_angle_raw = atan2f(z, x);
    float femur_distance = norm3df(z, x, 0);

    bool positiv_side = !signbit(z);
    const float* saturated_femur_of_interest =
        (positiv_side) ? dim.positiv_saturated_femur : dim.negativ_saturated_femur;
    const float& tibia_saturation =
        (positiv_side) ? dim.max_angle_tibia : -dim.min_angle_tibia;
    const float& femur_saturation =
        (positiv_side) ? dim.max_angle_femur : -dim.min_angle_femur;

    float tibia_angle_raw =
        atan2f(z - saturated_femur_of_interest[1], x - saturated_femur_of_interest[0]);
    if (!positiv_side) {
        tibia_angle_raw *= -1;
        femur_angle_raw *= -1;
    }

    bool too_close = femur_distance < dim.min_femur_to_gripper_dist;

    bool femur_condition = (femur_angle_raw <= femur_saturation);
    bool femur_cond_wider =
        femur_angle_raw <= (femur_saturation + dim.femur_overmargin_negative);
    bool tibia_condition =
        (((tibia_angle_raw <= (tibia_saturation + femur_saturation)) and
          !signbit(tibia_angle_raw)) or
         ((tibia_angle_raw <= (tibia_saturation + femur_saturation - 2 * pIgpu))));

    float center[2] = {0, 0};
    float radius = dim.min_femur_to_gripper_dist;
    bool& clamp_direction = too_close;

    if (femur_condition and not too_close) { // implies femur is not saturated,
        // and point is not in the innermost circle, so we clamp onto the outermost circle
        radius = dim.max_femur_to_gripper_dist;
    } else if (tibia_condition and not too_close) { // implies tibia is not saturated
        // and point is not  is the innercircle
        // so we clamp on the  outer winglet
        center[0] = saturated_femur_of_interest[0];
        center[1] = saturated_femur_of_interest[1];
        radius = dim.tibia_length;
    } else if (not femur_cond_wider and not tibia_condition) {
        // implies we are not between the two winglets and tibia is saturated
        center[0] = cosf(dim.femur_overmargin_negative + femur_saturation) *
                    dim.min_femur_to_gripper_dist;
        center[1] = sinf(dim.femur_overmargin_negative + femur_saturation) *
                    dim.min_femur_to_gripper_dist;
        center[1] = copysignf(center[1], z);
        radius = 0;
        clamp_direction = false;
    };

    clamp_on_circle(center, radius, clamp_direction, x, z);
}

__device__ __forceinline__ float3 dist_double_solf3(const float3& point,
                                                    const LegDimensions& dim) {
    float3 closest = point;
    place_over_coxa(closest, dim);
    float3 closest_flip = closest;

    float coxangle = find_coxa_angle(closest);
    float coxangle_flip = (coxangle > 0) ? coxangle - pIgpu : coxangle + pIgpu;

    finish_finding_closest(closest, dim, coxangle);
    finish_finding_closest(closest_flip, dim, coxangle_flip);

    float3* result_to_use = (norm3df(closest.x, closest.y, closest.z) <
                             norm3df(closest_flip.x, closest_flip.y, closest_flip.z))
                                ? &closest
                                : &closest_flip;

    return *result_to_use;
}

__device__ bool reachability_vect(const float3& point, const LegDimensions& dim)
// angle flipping on negative x values
{
    // Coxa as the frame of reference without rotation
    float3 result;
    result = point;
    place_over_coxa(result, dim);
    bool flip_flag = -signbit(result.x);

    if (flip_flag) {
        result.x *= -1;
        result.y *= -1;
    }

    // finding coxa angle
    float required_angle_coxa = find_coxa_angle(result);

    if (flip_flag) {
        result.x *= -1;
        result.y *= -1;
    }

    // flipping angle if above +-90deg

    if ((required_angle_coxa > dim.max_angle_coxa) ||
        (required_angle_coxa < dim.min_angle_coxa)) {
        return false;
    }

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox;
    float sin_angle_cox;
    cancel_coxa_rotation(result, required_angle_coxa, cos_angle_cox, sin_angle_cox);

    // Femur as the frame of reference witout rotation
    result.x -= dim.coxa_length;

    float linnorm = norm3df(result.x, result.y, result.z);

    if ((linnorm < dim.min_femur_to_gripper_dist) ||
        (linnorm > dim.max_femur_to_gripper_dist)) {
        return false;
    }

    // finding femur angle
    float required_angle_femur = atan2f(result.z, result.x);

    if ((required_angle_femur >= dim.min_angle_femur) &&
        (required_angle_femur <= dim.max_angle_femur)) {
        return true;
    }

    // distance to femur at the most extrem position, this value is pre_computed
    linnorm = fminf(norm3df(result.x - dim.positiv_saturated_femur[0], 0,
                            result.z - dim.positiv_saturated_femur[1]),
                    norm3df(result.x - dim.negativ_saturated_femur[0], 0,
                            result.z - dim.negativ_saturated_femur[1]));

    return linnorm <= dim.tibia_length;
}

__device__ bool reachability_absolute_tibia_limit(const float3& point,
                                                  const LegDimensions& dim)
// the tibia cannot exceed a specified cangle relative to the BODY
// this ensures that the tibia is for example always pointing down
{
    // Coxa as the frame of reference without rotation
    float3 result;
    result = point;
    place_over_coxa(result, dim);
    float required_angle_coxa;
    {
        bool flip_flag = -signbit(result.x);

        if (flip_flag) {
            result.x *= -1;
            result.y *= -1;
        }

        // finding coxa angle
        required_angle_coxa = find_coxa_angle(result);

        if (flip_flag) {
            result.x *= -1;
            result.y *= -1;
        }
    }

    if ((required_angle_coxa > dim.max_angle_coxa) ||
        (required_angle_coxa < dim.min_angle_coxa)) {
        return false;
    }

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox;
    float sin_angle_cox;
    cancel_coxa_rotation(result, required_angle_coxa, cos_angle_cox, sin_angle_cox);

    // Femur as the frame of reference witout rotation
    result.x -= dim.coxa_length;

    {
        float linnorm = norm3df(result.x, result.y, result.z);

        if ((linnorm < dim.min_femur_to_gripper_dist) ||
            (linnorm > dim.max_femur_to_gripper_dist)) {
            return false;
        }
    }

    bool in_negativ_tib_circle;
    bool in_positive_sat_circle;
    bool in_negative_sat_circle;
    bool inside_femur;
    bool in_positiv_tib_circle;

    {
        float deported_center_negativ[2];
        sincosf(dim.tibia_absolute_neg, &sin_angle_cox, &cos_angle_cox);
        deported_center_negativ[0] = dim.tibia_length * cos_angle_cox;
        deported_center_negativ[1] = dim.tibia_length * sin_angle_cox;
        // should not be inside
        in_negativ_tib_circle =
            norm3df(result.x - deported_center_negativ[0], 0,
                    result.z - deported_center_negativ[1]) <= dim.femur_length;
    }

    // finding femur angle
    {
        float required_angle_femur = atan2f(result.z, result.x);
        inside_femur = (required_angle_femur >= dim.min_angle_femur) &&
                       (required_angle_femur <= dim.tibia_absolute_pos);
    }
    {
        // - saturation is alowed
        in_negative_sat_circle =
            norm3df(result.x - dim.negativ_saturated_femur[0], 0,
                    result.z - dim.negativ_saturated_femur[1]) <= dim.tibia_length;
    }

    // distance to femur at the most extrem  femur position should be less than
    // tibia length to be in the saturation circle, circle center is
    // pre_computed

    {
        // We don´t want + saturation
        in_positive_sat_circle =
            norm3df(result.x - dim.positiv_saturated_femur[0], 0,
                    result.z - dim.positiv_saturated_femur[1]) <= dim.tibia_length;

        float deported_center_positiv[2];
        sincosf(dim.tibia_absolute_pos, &sin_angle_cox, &cos_angle_cox);
        deported_center_positiv[0] = dim.tibia_length * cos_angle_cox;
        deported_center_positiv[1] = dim.tibia_length * sin_angle_cox;
        // need it inside
        in_positiv_tib_circle =
            norm3df(result.x - deported_center_positiv[0], 0,
                    result.z - deported_center_positiv[1]) <= dim.femur_length;
    }

    bool reachability;
    reachability = (((!in_negativ_tib_circle) && (!in_positive_sat_circle)) &&
                    (in_negative_sat_circle || inside_femur || in_positiv_tib_circle));
    return reachability;
}

__global__ void dist_kernel(const Array<float3> input, const LegDimensions dimensions,
                            Array<float3> const output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] = dist_double_solf3(input.elements[i], dimensions);
        // output.elements[i] = input.elements[i];
    }
}

__global__ void reachability_kernel(const Array<float3> input,
                                    const LegDimensions dimensions,
                                    Array<bool> const output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] = reachability_vect(input.elements[i], dimensions);
    }
}

__global__ void reachability_abs_tib_kernel(const Array<float3> input,
                                            const LegDimensions dimensions,
                                            Array<bool> const output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] =
            reachability_absolute_tibia_limit(input.elements[i], dimensions);
    }
}

