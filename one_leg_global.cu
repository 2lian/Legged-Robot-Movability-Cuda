#include "hip/hip_runtime.h"
#include "octree_util.cu.h"
#include "one_leg.cu"
#include "settings.h"

__device__ __forceinline__ void z_rotateInPlace(float3& point, float z_rot,
                                                float& cos_memory, float& sin_memory) {
    sincosf(z_rot, &sin_memory, &cos_memory);
    float buffer = point.x * sin_memory;
    point.x = point.x * cos_memory - point.y * sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

__device__ __forceinline__ void z_unrotateInPlace(float3& point, float& cos_memory,
                                                  float& sin_memory) {
    float buffer = point.x * -sin_memory;
    point.x = point.x * cos_memory - point.y * -sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

/**
 * @brief will change the tibia_absolute_pos/neg according to the orientation
 *
 * @param quat
 * @param leg
 * @return new legdim
 */
__forceinline__ __device__ LegDimensions rotate_leg_data(Quaternion quat,
                                                         LegDimensions leg) {
    Quaternion quatOfLegAzimut = quatFromVectAngle(make_float3(0, 0, 1), leg.body_angle);

    Quaternion result =
        qtMultiply(qtMultiply(quatOfLegAzimut, quat), qtInvert(quatOfLegAzimut));
    float3 rpy = rpyFromQuat(result);
    float pitch = rpy.y;

    leg.tibia_absolute_pos -= pitch;
    leg.tibia_absolute_neg -= pitch;
    return leg;
}

__forceinline__ __device__ float3 make_asif_leg0(float3 point, LegDimensions leg,
                                                 float& cos_memory, float& sin_memory) {
    z_rotateInPlace(point, -leg.body_angle, cos_memory, sin_memory);
    return point;
}
__forceinline__ __device__ float3 undo_asif_leg0(float3 point, float cos_memory,
                                                 float sin_memory) {
    z_unrotateInPlace(point, cos_memory, sin_memory);
    return point;
}

template <typename Tout = bool, // function for distance
          Tout (*reach_function)(float3&, const LegDimensions&) = distance_circles>
__device__ __forceinline__ Tout distance_global(float3& point, const LegDimensions& dim,
                                                const Quaternion quat) {
    LegDimensions oriented_leg_dim = rotate_leg_data(quat, dim);
    // __shared__ LegDimensions oriented_leg_dim;
    // if (threadIdx.x == 0) {
        // oriented_leg_dim = rotate_leg_data(quat, dim);
    // }
    auto unrotated_point = qtInvRotate(quat, point);
    float cos_memory;
    float sin_memory;
    // __syncthreads();
    auto point_as_leg0 =
        make_asif_leg0(unrotated_point, oriented_leg_dim, cos_memory, sin_memory);
    Tout result = reach_function(point_as_leg0, oriented_leg_dim);
    auto rerotated_point = undo_asif_leg0(point_as_leg0, cos_memory, sin_memory);
    rerotated_point = qtRotate(quat, rerotated_point);
    point = rerotated_point;
    return result;
}

template <typename Tout = bool, // function for reachability
          Tout (*reach_function)(const float3&,
                                 const LegDimensions&) = reachability_circles>
__device__ __forceinline__ Tout reachability_global(const float3& point,
                                                    const LegDimensions& dim,
                                                    const Quaternion quat) {
    LegDimensions oriented_leg_dim = rotate_leg_data(quat, dim);
    // __shared__ LegDimensions oriented_leg_dim;
    // if (threadIdx.x == 0) {
        // oriented_leg_dim = rotate_leg_data(quat, dim);
    // }}
    auto unrotated_point = qtInvRotate(quat, point);
    float cos_memory;
    float sin_memory;
    // __syncthreads();
    auto point_as_leg0 =
        make_asif_leg0(unrotated_point, oriented_leg_dim, cos_memory, sin_memory);
    Tout result = reach_function(point_as_leg0, oriented_leg_dim);
    return result;
}

// constexpr Quaternion quat = {1, 0, 0, 0};
// constexpr Quaternion quat = {0.999, 0, 0.01, 0};
// constexpr Quaternion quat = {0.996, 0, -0.087, 0};
constexpr Quaternion quat = {0.985, 0, 0.174, 0};
// constexpr Quaternion quat = {0.924, 0, -0.384, 0}; // y rot 40deg
// constexpr Quaternion quat = {0.940, 0, 0, 0.342}; // z rot 40deg
__global__ void reachability_global_kernel(const Array<float3> input,
                                           const LegDimensions dim, Array<bool> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] = reachability_global(input.elements[i], dim, quat);
    }
}
__global__ void distance_global_kernel(const Array<float3> input, const LegDimensions dim,
                                       Array<float3> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        float3 result = input.elements[i];
        bool reachability = distance_global(result, dim, quat);
        output.elements[i] = result;
    }
}

__device__ __forceinline__ float3 centerPoint(float3 p1, float3 p2) {
    float3 center;
    center.x = p1.x + p2.x;
    center.y = p1.y + p2.y;
    center.z = p1.z + p2.z;
    return center;
}

// constexpr hipStream_t streams[10];
__global__ void recursive_kernel(Box box, const Array<float3> input,
                                 const LegDimensions leg, Array<float3> output,
                                 uchar depth, float rad, bool validity) {

    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    uchar quadCount = 3;
    bool small[3];
    small[0] = abs(box.topOffset.x) < MIN_BOX_X;
    small[1] = abs(box.topOffset.y) < MIN_BOX_Y;
    small[2] = abs(box.topOffset.z) < MIN_BOX_Z;
    if (small[0])
        quadCount -= 1;
    if (small[1])
        quadCount -= 1;
    if (small[2])
        quadCount -= 1;

    // quad_count = 3;
    const uint max_quad_ind = pow(pow(2, quadCount), SUB_QUAD);
    for (uint computeIndex = index; computeIndex < max_quad_ind; computeIndex += stride) {

        Box new_box;
        uchar missingQuad;
        CreateChildBox(box, new_box, quadCount, computeIndex, small, missingQuad);

        if (missingQuad == DEADQUADRAN)
            continue;

        auto subQuadCount = 3 - missingQuad;
        bool tooSmall = subQuadCount <= 0;
        bool notCloseEnough;
        float3 distToNewBox;
        if constexpr (SUB_QUAD <= 1) {
            notCloseEnough = false;
        } else {
            distToNewBox =
                maxi(abs(new_box.center - box.center) + abs(new_box.topOffset), 0);
            notCloseEnough = (linorm(distToNewBox) < rad);
        }

        auto distance = new_box.center;
        // distance.y = 0.1;
        bool reachabilityEdgeInTheBox = not notCloseEnough;
        bool reachability = validity;
        if (reachabilityEdgeInTheBox) {
            reachability = distance_global(distance, leg, quat);
            reachabilityEdgeInTheBox = linorm(distance) < linorm(new_box.topOffset);
        } else
            distance = make_float3(linorm(distToNewBox) - rad, 0, 0);

        auto radius = linorm(distance);
        if (reachabilityEdgeInTheBox and (not tooSmall) and (depth < MAX_DEPTH)) {
            // auto subDistance =
            uint childCount = pow(pow(2, subQuadCount), SUB_QUAD);
            constexpr uint maxBlockSize = 1024 / 4;
            int blockSize = min(childCount, maxBlockSize);
            int numBlock = (childCount + blockSize - 1) / blockSize;
            // numBlock = 1;
            // blockSize = min(childCount, 24);
            // blockSize = 24;
            auto validity = reachability;
            recursive_kernel<<<numBlock, blockSize>>>(new_box, input, leg, output,
                                                      depth + 1, radius, validity);
        } else if constexpr (OutputOctree) {
            // distance = make_float3(min(radius, (float)500), 0, 0);
            distance = make_float3(depth, 0, 0);
            // distance = make_float3((threadIdx.x/24) + 0.01, 0, 0);
            if (tooSmall) {
                // distance = distance * 0;
            }
            if (depth == MAX_DEPTH) {
                // distance = make_float3(0,0,0);
            }
            if (reachability and not reachabilityEdgeInTheBox) {
                // distance = make_float3(-1,0,0);
            }

            constexpr int blockSize = 1024 / 4;
            int numBlock = (input.length + blockSize - 1) / blockSize;
            fillOutKernel<<<numBlock, blockSize>>>(new_box, distance, input, output);
        }
    }
}

__device__ bool distance(float3& point, const LegDimensions& dim, const Quaternion quat) {
    return distance_global(point, dim, quat);
}
