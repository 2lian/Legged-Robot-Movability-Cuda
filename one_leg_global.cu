#include "hip/hip_runtime.h"
#include "octree_util.cu.h"
#include "one_leg.cu"
#include "settings.h"
#include "thrust/detail/tuple.inl"
#include "thrust/tuple.h"
#include "unified_math_cuda.cu.h"
#include <iostream>
#include <ostream>

__device__ __forceinline__ thrust::tuple<float3, bool>
simpleGravityCheck(const float3 target, const Quaternion orientation, LegDimensions dim) {
    float3 coxaJoint =
        make_float3(cos(dim.body_angle) * dim.body, sin(dim.body_angle) * dim.body, 0);
    float3 rotatedCoxa = qtRotate(orientation, coxaJoint);
    float3 oriToCoxaHoriz = rotatedCoxa * make_float3(1, 1, 0);
    float3 coxaOrigTarget = target - coxaJoint;
    float dotprod = dot(coxaOrigTarget, oriToCoxaHoriz);
    bool validity = dotprod > 0;
    float3 planeToPoint = oriToCoxaHoriz * dotprod / linormRaw(oriToCoxaHoriz);
    // float3 projection = coxaOrigTarget - oriToCoxaHoriz *
    // dot(coxaOrigTarget, oriToCoxaHoriz) /
    // linormRaw(oriToCoxaHoriz);
    return thrust::make_tuple(planeToPoint, validity);
}

__host__ __device__ __forceinline__ void
z_rotateInPlace(float3& point, float z_rot, float& cos_memory, float& sin_memory) {
    sincosf(z_rot, &sin_memory, &cos_memory);
    float buffer = point.x * sin_memory;
    point.x = point.x * cos_memory - point.y * sin_memory;
    point.y = buffer + point.y * cos_memory;
}

__host__ __device__ __forceinline__ void
z_unrotateInPlace(float3& point, float& cos_memory, float& sin_memory) {
    float buffer = point.x * -sin_memory;
    point.x = point.x * cos_memory - point.y * -sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

/**
 * @brief will change the tibia_absolute_pos/neg according to the orientation
 *
 * @param quat
 * @param leg
 * @return new legdim
 */
__forceinline__ __host__ __device__ LegDimensions rotate_leg_data(Quaternion quat,
                                                                  LegDimensions leg) {
    Quaternion quatOfLegAzimut = quatFromVectAngle(make_float3(0, 0, 1), leg.body_angle);

    Quaternion result =
        qtMultiply(qtMultiply(quatOfLegAzimut, quat), qtInvert(quatOfLegAzimut));
    float3 rpy = rpyFromQuat(result);
    float pitch = rpy.y;

    leg.tibia_absolute_pos -= pitch;
    leg.tibia_absolute_neg -= pitch;
    return leg;
}

__forceinline__ __host__ __device__ float3 make_asif_leg0(float3 point, LegDimensions leg,
                                                          float& cos_memory,
                                                          float& sin_memory) {
    z_rotateInPlace(point, -leg.body_angle, cos_memory, sin_memory);
    return point;
}
__forceinline__ __device__ float3 undo_asif_leg0(float3 point, float cos_memory,
                                                 float sin_memory) {
    z_unrotateInPlace(point, cos_memory, sin_memory);
    return point;
}

template <typename Tout = bool, // function for distance
          Tout (*reach_function)(float3&, const LegDimensions&) = distance_circles>
__device__ __forceinline__ Tout distance_global(float3& point, const LegDimensions& dim,
                                                const Quaternion quat) {
    // LegDimensions oriented_leg_dim = rotate_leg_data(quat, dim);
    __shared__ LegDimensions oriented_leg_dim;
    if (threadIdx.x == 0) {
        oriented_leg_dim = rotate_leg_data(quat, dim);
    }
    auto unrotated_point = qtInvRotate(quat, point);
    float cos_memory;
    float sin_memory;
    __syncthreads();
    auto point_as_leg0 =
        make_asif_leg0(unrotated_point, oriented_leg_dim, cos_memory, sin_memory);
    Tout result = reach_function(point_as_leg0, oriented_leg_dim);
    auto rerotated_point = undo_asif_leg0(point_as_leg0, cos_memory, sin_memory);
    rerotated_point = qtRotate(quat, rerotated_point);
    point = rerotated_point;
    return result;
}

template <typename Tout = bool, // function for reachability
          Tout (*reach_function)(const float3&,
                                 const LegDimensions&) = reachability_circles>
__host__ __device__ __forceinline__ Tout reachability_global(const float3& point,
                                                             const LegDimensions& dim,
                                                             const Quaternion quat) {
#ifdef __CUDA_ARCH__
    // LegDimensions oriented_leg_dim = rotate_leg_data(quat, dim);
    __shared__ LegDimensions oriented_leg_dim;
    if (threadIdx.x == 0) {
        oriented_leg_dim = rotate_leg_data(quat, dim);
    }
#else
    LegDimensions oriented_leg_dim = rotate_leg_data(quat, dim);
#endif

    auto unrotated_point = qtInvRotate(quat, point);
    float cos_memory;
    float sin_memory;

#ifdef __CUDA_ARCH__
    __syncthreads();
#endif
    auto point_as_leg0 =
        make_asif_leg0(unrotated_point, oriented_leg_dim, cos_memory, sin_memory);
    Tout result = reach_function(point_as_leg0, oriented_leg_dim);
    return result;
}

__host__ void reachability_kernel_cpu(const Array<float3> input, const LegDimensions dim,
                                      Array<bool> output) {
    for (int i = 0; i < input.length; i += 0) {
        std::cout << "hey" << std::endl;
        output.elements[i] = reachability_global(input.elements[i], dim, quatTest);
    }
}

__global__ void reachability_global_kernel(const Array<float3> input,
                                           const LegDimensions dim, Array<bool> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] = reachability_global(input.elements[i], dim, quatTest);
    }
}
__global__ void distance_global_kernel(const Array<float3> input, const LegDimensions dim,
                                       Array<float3> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        float3 result = input.elements[i];
        bool reachability = distance_global(result, dim, quatTest);
        output.elements[i] = result;
    }
}

__global__ void recursive_kernel(Box box, const Array<float3> input,
                                 const LegDimensions leg, Array<float3> output,
                                 uchar depth, float rad, bool validity) {

    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    uchar quadCount = 3;
    bool small[3];
    small[0] = abs(box.topOffset.x) < MIN_BOX_X;
    small[1] = abs(box.topOffset.y) < MIN_BOX_Y;
    small[2] = abs(box.topOffset.z) < MIN_BOX_Z;
    if (small[0])
        quadCount -= 1;
    if (small[1])
        quadCount -= 1;
    if (small[2])
        quadCount -= 1;

    // quad_count = 3;
    const uint max_quad_ind = pow(pow(2, quadCount), SUB_QUAD);
    for (uint computeIndex = index; computeIndex < max_quad_ind; computeIndex += stride) {

        Box new_box;
        uchar missingQuad;
        CreateChildBox(box, new_box, quadCount, computeIndex, small, missingQuad);

        if (missingQuad == DEADQUADRAN)
            continue;

        auto subQuadCount = 3 - missingQuad;
        bool tooSmall = subQuadCount <= 0;
        bool notCloseEnough;
        float3 distToNewBox;
        if constexpr (SUB_QUAD <= 1) {
            notCloseEnough = false;
        } else {
            distToNewBox =
                maxi(abs(new_box.center - box.center) + abs(new_box.topOffset), 0);
            notCloseEnough = (linorm(distToNewBox) < rad);
        }

        auto distance = new_box.center;
        // distance.y = 0.1;
        bool reachabilityEdgeInTheBox = not notCloseEnough;
        bool reachability = validity;
        if (reachabilityEdgeInTheBox) {
            reachability = distance_global(distance, leg, quatTest);
            reachabilityEdgeInTheBox = linorm(distance) < linorm(new_box.topOffset);
        } else
            distance = make_float3(linorm(distToNewBox) - rad, 0, 0);

        auto radius = linorm(distance);
        if (reachabilityEdgeInTheBox and (not tooSmall) and (depth < MAX_DEPTH)) {
            // auto subDistance =
            uint childCount = pow(pow(2, subQuadCount), SUB_QUAD);
            constexpr uint maxBlockSize = 1024 / 4;
            int blockSize = min(childCount, maxBlockSize);
            int numBlock = (childCount + blockSize - 1) / blockSize;
            // numBlock = 1;
            // blockSize = min(childCount, 24);
            // blockSize = 24;
            auto validity = reachability;
            recursive_kernel<<<numBlock, blockSize>>>(new_box, input, leg, output,
                                                      depth + 1, radius, validity);
        } else if constexpr (OutputOctree) {
            // distance = make_float3(min(radius, (float)500), 0, 0);
            distance = make_float3(depth, 0, 0);
            // distance = make_float3((threadIdx.x/24) + 0.01, 0, 0);
            if (tooSmall) {
                // distance = distance * 0;
            }
            if (depth == MAX_DEPTH) {
                // distance = make_float3(0,0,0);
            }
            if (reachability and not reachabilityEdgeInTheBox) {
                // distance = make_float3(-1,0,0);
            }

            constexpr int blockSize = 1024 / 4;
            int numBlock = (input.length + blockSize - 1) / blockSize;
            fillOutKernel<<<numBlock, blockSize>>>(new_box, distance, input, output);
        }
    }
}

__device__ bool distance(float3& point, const LegDimensions& dim, const Quaternion quat) {
    if constexpr (enableGrav) {
        const auto res = simpleGravityCheck(point, quat, dim);
        bool gravValid = thrust::get<1>(res);
        float3 gravDist = thrust::get<0>(res);
        if (not gravValid) {
            point = gravDist;
            return false;
        }
    }
    return distance_global(point, dim, quat);
}
