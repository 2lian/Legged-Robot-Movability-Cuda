#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "Header.h"

RobotDimensions dim_of_SCARE() {
    struct RobotDimensions scare{};

    scare.pI = 3.141592653589793238462643383279502884197169f;
    scare.body = 185.0f;
    scare.coxa_angle_deg = 60.0f;
    scare.coxa_length = 165.0f;
    scare.tibia_angle_deg = 120.0f; //90
    scare.tibia_length = 190.0f;
    scare.tibia_length_squared = scare.tibia_length * scare.tibia_length;
    scare.femur_angle_deg = 150.0f; //120
    scare.femur_length = 300.0f; //200
    scare.max_angle_coxa = scare.pI / 180.0f * scare.coxa_angle_deg;
    scare.min_angle_coxa = -scare.pI / 180.0f * scare.coxa_angle_deg;
    scare.max_angle_coxa_w_margin = scare.pI / 180.0f * (scare.coxa_angle_deg - 10.0f);
    scare.min_angle_coxa_w_margin = -scare.pI / 180.0f * (scare.coxa_angle_deg - 10.0f);
    scare.max_angle_tibia = scare.pI / 180.0f * scare.tibia_angle_deg;
    scare.min_angle_tibia = -scare.pI / 180.0f * scare.tibia_angle_deg;
    scare.max_angle_femur = scare.max_angle_tibia;
    scare.min_angle_femur = scare.min_angle_tibia;
    scare.max_angle_femur_w_margin = scare.pI / 180.0f * (scare.tibia_angle_deg + 20.0f);
    scare.min_angle_femur_w_margin = -scare.pI / 180.0f * (scare.tibia_angle_deg + 20.0f);
    scare.max_tibia_to_gripper_dist = scare.tibia_length + scare.femur_length;

    scare.positiv_saturated_femur[0] = cos(scare.max_angle_femur) * scare.femur_length;
    scare.positiv_saturated_femur[1] = sin(scare.max_angle_femur) * scare.femur_length;

    scare.negativ_saturated_femur[0] = cos(scare.min_angle_femur) * scare.femur_length;
    scare.negativ_saturated_femur[1] = sin(scare.min_angle_femur) * scare.femur_length;

    scare.fem_tib_min_host[0] = scare.tibia_length + scare.femur_length * cos(scare.pI / 180.0f * scare.femur_angle_deg);
    scare.fem_tib_min_host[1] = scare.femur_length * sin(scare.pI / 180.0f * scare.femur_angle_deg);

    scare.min_tibia_to_gripper_dist = sqrt(scare.fem_tib_min_host[0] * scare.fem_tib_min_host[0]
                                           + scare.fem_tib_min_host[1] * scare.fem_tib_min_host[1]);
    scare.middle_TG = (scare.max_tibia_to_gripper_dist + scare.min_tibia_to_gripper_dist) / 2.0f;

    return scare;
}

__global__
void empty_kernel() {
}

// Function to calculate the mean of an array of floats
float calculateMean(const float* arr, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; ++i) {
        sum += arr[i];
    }
    return sum / size;
}

// Function to calculate the standard deviation of an array of floats
float calculateStdDev(const float* arr, int size, float mean) {
    float sum = 0.0f;
    for (int i = 0; i < size; ++i) {
        float diff = arr[i] - mean;
        sum += diff * diff;
    }
    return std::sqrt(sum / (size - 1));
}

__device__ float sumOfSquares3df(const float* vector) {
    return vector[0] * vector[0] + vector[1] * vector[1] + vector[2] * vector[2];
}

__device__
void dist_noflip(float* point, RobotDimensions& dim, float* result_point)
// no angle flipping
{
    // Coxa as the frame of reference without rotation
    float result[3];
    result[0] = point[0] - dim.body;
    result[1] = point[1];
    result[2] = point[2];

    // finding coxa angle
    float required_angle_coxa = atan2f(result[1], result[0]);

    // flipping angle if above +-90deg
    // required_angle_coxa = fmodf(required_angle_coxa + dim.pI / 2.f + 2.f * dim.pI, dim.pI) - dim.pI / 2.f;

    // saturating coxa angle for dist
    required_angle_coxa = fmaxf(fminf(required_angle_coxa, dim.max_angle_coxa_w_margin), dim.min_angle_coxa_w_margin);

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox = cosf(-required_angle_coxa);
    float sin_angle_cox = sinf(-required_angle_coxa);
    float buffer = result[0] * sin_angle_cox;
    result[0] = result[0] * cos_angle_cox - result[1] * sin_angle_cox;
    result[1] = buffer + result[1] * cos_angle_cox;

    // Femur as the frame of reference witout rotation
    result[0] -= dim.coxa_length;

    // finding femur angle
    float required_angle_femur = atan2f(result[2], result[0]);

    // saturating coxa angle for dist
    required_angle_femur = fmaxf(fminf(required_angle_femur, dim.max_angle_femur_w_margin), dim.min_angle_femur_w_margin);

    // canceling femur rotation for dist
    float cos_angle_fem = cosf(required_angle_femur);
    float sin_angle_fem = sinf(required_angle_femur);

    // middle_TG as the frame of reference
    result[0] -= dim.middle_TG * cos_angle_fem;
    result[2] -= dim.middle_TG * sin_angle_fem;

    // rotating back to default xyz, but staying on middle_TG

    buffer = result[1] * sin_angle_cox;
    result[1] = -result[0] * sin_angle_cox + result[1] * cos_angle_cox;
    result[0] = result[0] * cos_angle_cox + buffer;

    result_point[0] = result[0];
    result_point[1] = result[1];
    result_point[2] = result[2];

    return;
}

__device__
void dist_flip(float* point, RobotDimensions& dim, float* result_point)
// with angle flipping
{
    // Coxa as the frame of reference without rotation
    float result[3];
    result[0] = point[0] - dim.body;
    result[1] = point[1];
    result[2] = point[2];

    // finding coxa angle
    float required_angle_coxa = atan2f(-result[1], -result[0]);

    // flipping angle if above +-90deg
    // required_angle_coxa = fmodf(required_angle_coxa + dim.pI / 2.f + 2.f * dim.pI, dim.pI) - dim.pI / 2.f;

    // saturating coxa angle for dist
    required_angle_coxa = fmaxf(fminf(required_angle_coxa, dim.max_angle_coxa_w_margin), dim.min_angle_coxa_w_margin);

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox = cosf(-required_angle_coxa);
    float sin_angle_cox = sinf(-required_angle_coxa);
    float buffer = result[0] * sin_angle_cox;
    result[0] = result[0] * cos_angle_cox - result[1] * sin_angle_cox;
    result[1] = buffer + result[1] * cos_angle_cox;

    // Femur as the frame of reference witout rotation
    result[0] -= dim.coxa_length;

    // finding femur angle
    float required_angle_femur = atan2f(result[2], result[0]);

    // saturating coxa angle for dist
    required_angle_femur = fmaxf(fminf(required_angle_femur, dim.max_angle_femur_w_margin), dim.min_angle_femur_w_margin);

    // canceling femur rotation for dist
    float cos_angle_fem = cosf(required_angle_femur);
    float sin_angle_fem = sinf(required_angle_femur);

    // middle_TG as the frame of reference
    result[0] -= dim.middle_TG * cos_angle_fem;
    result[2] -= dim.middle_TG * sin_angle_fem;

    // rotating back to default xyz, but staying on middle_TG

    buffer = result[1] * sin_angle_cox;
    result[1] = -result[0] * sin_angle_cox + result[1] * cos_angle_cox;
    result[0] = result[0] * cos_angle_cox + buffer;

    result_point[0] = result[0];
    result_point[1] = result[1];
    result_point[2] = result[2];

    return;
}

__device__
void dist_double_sol(float* point, RobotDimensions& dim, float* result_point)
//
{
    // Coxa as the frame of reference without rotation
    float result_noflip[3];
    float result_flip[3];

    dist_noflip(point, dim, result_noflip);
    dist_flip(point, dim, result_flip);

    float* result_to_use = (sumOfSquares3df(result_noflip) < sumOfSquares3df(result_flip)) ? result_noflip : result_flip;
    // result_to_use = result_flip;
    result_point[0] = result_to_use[0];
    result_point[1] = result_to_use[1];
    result_point[2] = result_to_use[2];

}

// Kernel launch function
__global__
void dist_kernel(Matrix table, RobotDimensions dimensions, Matrix result_table)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        dist_double_sol((table.elements + i * table.width), dimensions, (result_table.elements + i * result_table.width));
    }
}

__global__
void norm3df_kernel(Matrix table, Matrix result_table)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        result_table.elements[i] = norm3df(table.elements[i * table.width],
                                           table.elements[i * table.width+1],
                                           table.elements[i * table.width+2]);
    }
}

__global__
void change_z_kernel(Matrix table, float zval)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        table.elements[i * table.width+2] = zval;
    }
}

AutoEstimator::AutoEstimator(int pxWidth, int pxHeight) {
    blockSize = 1024;
    verbose = true;
    screenWidth = pxWidth;
    screenHeight = pxHeight;
    dimensions = dim_of_SCARE();
    rows = screenWidth * screenHeight;

    table_input.width = 3; table_input.height = rows;
    table_input.elements = new float[table_input.width * table_input.height];

    result.width = 3; result.height = rows;
    result.elements = new float[table_input.width * table_input.height];

    result_norm.width = 1; result_norm.height = rows;
    result_norm.elements = new float[table_input.width * table_input.height];

    table_input_gpu.width = table_input.width; table_input_gpu.height = table_input.height;
    result_gpu.width = table_input_gpu.width; result_gpu.height = table_input_gpu.height;
    result_norm_gpu.width = result_norm.width; result_norm_gpu.height = table_input_gpu.height;

    numBlocks = (rows + blockSize - 1) / blockSize;
    error_check();
    setup_kernel();
    input_as_grid();
    alocate_gpu_mem();
    copy_input_cpu2gpu();
}

void AutoEstimator::error_check(){
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error or exit the program as needed.
    }
}

void AutoEstimator::input_as_grid(){
    for (int i = 0; i < screenHeight; i++) {
        for (int j = 0; j < screenWidth; j++) {
            int row = i * screenWidth + j;
            // X
            *(table_input.elements + row * table_input.width + 0)
                    //= -1000.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 2000.0f;
                    = (float)(j - screenWidth /2);
            // Y
            *(table_input.elements + row * table_input.width + 1)
                    //= -1000.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 2000.0f;
                    = -(float)(i - screenHeight / 2);
            // Z
            *(table_input.elements + row * table_input.width + 2)
                    //= -500.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 1000.0f;
                    = 0.f;
        }
    }
    if (verbose) { std::cout << "Host grid generated" << std::endl; }
}

void AutoEstimator::change_zvalue(float zvalue){

    change_z_kernel<<<numBlocks, blockSize >>>(table_input_gpu, zvalue);

}

void AutoEstimator::alocate_gpu_mem(){
    hipMalloc(&table_input_gpu.elements, table_input_gpu.width * table_input_gpu.height * sizeof(float));

    hipMalloc(&result_gpu.elements, result_gpu.width * result_gpu.height * sizeof(float));

    hipMalloc(&result_norm_gpu.elements, result_norm_gpu.width * result_norm_gpu.height * sizeof(float));

    if (verbose) { std::cout << "GPU memory allocated" << std::endl; }
    error_check();
}

void AutoEstimator::copy_input_cpu2gpu(){
    hipMemcpy(table_input_gpu.elements,
               table_input.elements,
               table_input.width * table_input.height * sizeof(float),
               hipMemcpyHostToDevice);
    if (verbose) { std::cout << "Host data copied to GPU" << std::endl; }
    error_check();
}

void AutoEstimator::setup_kernel(){
    std::wcout << "Threads per block: " << blockSize << "\nNumber of blocks: " << numBlocks << std::endl;
    empty_kernel<<<numBlocks, blockSize >>>();
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Empty kernels started" << std::endl;}
    error_check();
}

void AutoEstimator::compute_dist(){
    if (verbose) { std::cout << "Compute started" << std::endl;}
    dist_kernel<<<numBlocks, blockSize >>>(table_input_gpu, dimensions, result_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::compute_result_norm(){
    if (verbose) { std::cout << "Compute norm started" << std::endl;}
    norm3df_kernel<<<numBlocks, blockSize >>>(result_gpu, result_norm_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::copy_output_gpu2cpu(){
    hipMemcpy(result.elements,
               result_gpu.elements,
               result_gpu.width * result_gpu.height * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(result_norm.elements,
               result_norm_gpu.elements,
               result_norm_gpu.width * result_norm_gpu.height * sizeof(float),
               hipMemcpyDeviceToHost);
    if (verbose) { std::cout << "GPU result copied to host" << std::endl;}
    hipDeviceSynchronize();
    error_check();
}

void AutoEstimator::delete_all(){
    delete[] table_input.elements;
    hipFree(table_input_gpu.elements);
    hipFree(result_gpu.elements);
    hipFree(result_norm_gpu.elements);
    delete[] result.elements;
    delete[] result_norm.elements;
    if (verbose) { std::cout << "All pointers deleted" << std::endl;}
    error_check();
}

int main_le_old(void)
{
    AutoEstimator autoe{6, 7};
    std::cout << "initializing c++\n";
    //getting scare's dimensions in an object
    const RobotDimensions dimensions = dim_of_SCARE();

    // number of rows is the size of a 2K screen * 10
    int screenWidth = 1920;
    int screenHeight = 1080;
    int scale_up_factor = 1;
    int rows = screenWidth * screenHeight * scale_up_factor; // 1 << 20 is equivalent to 2,097,152.

    // Creates the matrix object (not a matrix but row/col are stored in row major) so:
    // M(row, col) = *(table_input.elements + row * table_input.width + col)
    Matrix table_input;
    table_input.width = 3; table_input.height = rows;
    table_input.elements = new float[table_input.width * table_input.height];

    // Initialize the vectorList with random values within -1000 1000
    for (int i = 0; i < screenHeight; i++) {
        for (int j = 0; j < screenWidth; j++) {
            int row = i * screenWidth + j;
            // X
            *(table_input.elements + row * table_input.width + 0)
                    //= -1000.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 2000.0f;
                    = (float)(j - screenWidth /2);
            // Y
            *(table_input.elements + row * table_input.width + 1)
                    //= -1000.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 2000.0f;
                    = -(float)(i - screenHeight / 2);
            // Z
            *(table_input.elements + row * table_input.width + 2)
                    //= -500.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 1000.0f;
                    = 200.f;
        }
    }
    // *(table_input.elements + 0 * table_input.width + 0) = 600.f;
    // *(table_input.elements + 0 * table_input.width + 1) = 0.f;
    // *(table_input.elements + 0 * table_input.width + 2) = 0.f;
    //
    // *(table_input.elements + 1 * table_input.width + 0) = 700.f;
    // *(table_input.elements + 1 * table_input.width + 1) = 0.f;
    // *(table_input.elements + 1 * table_input.width + 2) = 0.f;

    std::cout << "allocating GPU memory\n";

    // Allocating memory on the gpu for the input
    Matrix table_input_gpu;
    table_input_gpu.width = table_input.width; table_input_gpu.height = table_input.height;
    hipMalloc(&table_input_gpu.elements, table_input_gpu.width * table_input_gpu.height * sizeof(float));

    // and the output
    Matrix result_gpu;
    result_gpu.width = table_input_gpu.width; result_gpu.height = table_input_gpu.height;
    hipMalloc(&result_gpu.elements, result_gpu.width * result_gpu.height * sizeof(float));

    // Copying from  host to gpu (only for the input)
    hipMemcpy(table_input_gpu.elements, table_input.elements, table_input.width * table_input.height * sizeof(float), hipMemcpyHostToDevice);


    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run kernel on 1M elements on the GPU
    int blockSize = 1024; // threads per block (multiples of 32 are best)
    int numBlocks = (rows + blockSize - 1) / blockSize; // number of blocks
    std::wcout << "Threads per block: " << blockSize << "\nNumber of blocks: " << numBlocks << std::endl;
    // int blockSize = rows; // threads per block
    // int numBlocks = 1; // number of blocks

    // Add an "empty" kernel to incur setup overhead
    empty_kernel<<<numBlocks, blockSize >>>();

    std::cout << "Running batch of kernel\n";
    // Calculate execution time per row
    // for several runs
    int const runs = 50;
    float milliseconds;
    float executionTimePerRow;
    float time_vect[runs];
    float time_vect_per_row[runs];
    hipError_t cudaStatus;

    for (int index = 0; index < runs; ++index) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Record the start time
        hipEventRecord(start);

        //Compute starts
        dist_kernel<<<numBlocks, blockSize >>>(table_input_gpu, dimensions, result_gpu);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Record the end time
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        //extract time from GPU
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        //converts times and stores in the vector
        executionTimePerRow = milliseconds / (float)rows * 1000000.f;
        time_vect[index] = milliseconds;
        time_vect_per_row[index] = executionTimePerRow;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error or exit the program as needed.
    }

    float mean = calculateMean(time_vect, runs);

    std::cout << "\nTotal compute time for " << rows/1'000'000 << " M rows is " << mean << " ms on avg \nThat's " << 1.f / mean * 1e3 * (float)scale_up_factor << " Hz on a 2k screen" << "\n";

    // Calculate the average
    mean = calculateMean(time_vect_per_row, runs);
    float stddev = calculateStdDev(time_vect_per_row, runs, mean);

    // Calculate the standard deviation
    std::cout << "Execution time per row: " << mean << "ns\n(std.dev.of " << stddev << ")" << std::endl;

    // extract GPU memory onto host
    // Allocate
    float* result_host = new float[result_gpu.width * result_gpu.height];
    // Copy data from GPU to host
    hipMemcpy(result_host, result_gpu.elements, result_gpu.width* result_gpu.height * sizeof(float), hipMemcpyDeviceToHost);

    // ################################ WRITING TO BIN vvvvvvv
    // Specify the file path
    const char* binaryFilePath = "result.bin";

    // Open the binary file for writing
    std::ofstream outputFile(binaryFilePath, std::ios::binary);

    // Check if the file is open
    if (!outputFile.is_open()) {
        std::cerr << "Failed to open the binary file for writing." << std::endl;
    }
    else {
        // Write the data to the binary file
        outputFile.write(reinterpret_cast<const char*>(result_host), sizeof(float) * result_gpu.width * result_gpu.height);

        // Close the binary file
        outputFile.close();
        std::cout << "Data has been written to " << binaryFilePath << std::endl;
    }
    // ################################ WRITING TO BIN ^^^^

    // Free memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(table_input_gpu.elements);
    hipFree(result_gpu.elements);
    delete[] table_input.elements;
    delete[] result_host;

    return 0;
}