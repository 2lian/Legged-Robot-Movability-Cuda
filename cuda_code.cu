#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "Header.h"

RobotDimensions dim_of_SCARE() {
    struct RobotDimensions scare{};

    scare.pI = 3.141592653589793238462643383279502884197169f;
    scare.body = 185.0f;
    scare.coxa_angle_deg = 60.0f;
    scare.coxa_length = 165.0f;
    scare.tibia_angle_deg = 90.0f; //90
    scare.tibia_length = 190.0f;
    scare.tibia_length_squared = scare.tibia_length * scare.tibia_length;
    scare.femur_angle_deg = 120.0f; //120
    scare.femur_length = 200.0f; //200
    scare.max_angle_coxa = scare.pI / 180.0f * scare.coxa_angle_deg;
    scare.min_angle_coxa = -scare.pI / 180.0f * scare.coxa_angle_deg;
    scare.max_angle_coxa_w_margin = scare.pI / 180.0f * (scare.coxa_angle_deg - 10.0f);
    scare.min_angle_coxa_w_margin = -scare.pI / 180.0f * (scare.coxa_angle_deg - 10.0f);
    scare.max_angle_tibia = scare.pI / 180.0f * scare.tibia_angle_deg;
    scare.min_angle_tibia = -scare.pI / 180.0f * scare.tibia_angle_deg;
    scare.max_angle_femur = scare.max_angle_tibia;
    scare.min_angle_femur = scare.min_angle_tibia;
    scare.max_angle_femur_w_margin = scare.pI / 180.0f * (scare.tibia_angle_deg + 20.0f);
    scare.min_angle_femur_w_margin = -scare.pI / 180.0f * (scare.tibia_angle_deg + 20.0f);
    scare.max_tibia_to_gripper_dist = scare.tibia_length + scare.femur_length;

    scare.positiv_saturated_femur[0] = cos(scare.max_angle_femur) * scare.tibia_length;
    scare.positiv_saturated_femur[1] = sin(scare.max_angle_femur) * scare.tibia_length;

    scare.negativ_saturated_femur[0] = cos(scare.min_angle_femur) * scare.tibia_length;
    scare.negativ_saturated_femur[1] = sin(scare.min_angle_femur) * scare.tibia_length;

    scare.fem_tib_min_host[0] = scare.tibia_length + scare.femur_length * cos(scare.pI / 180.0f * scare.femur_angle_deg);
    scare.fem_tib_min_host[1] = scare.femur_length * sin(scare.pI / 180.0f * scare.femur_angle_deg);

    scare.min_tibia_to_gripper_dist = sqrt(scare.fem_tib_min_host[0] * scare.fem_tib_min_host[0]
                                           + scare.fem_tib_min_host[1] * scare.fem_tib_min_host[1]);
    scare.middle_TG = (scare.max_tibia_to_gripper_dist + scare.min_tibia_to_gripper_dist) / 2.0f;

    return scare;
}

__global__
void empty_kernel() {
}

// Function to calculate the mean of an array of floats
float calculateMean(const float* arr, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; ++i) {
        sum += arr[i];
    }
    return sum / size;
}

// Function to calculate the standard deviation of an array of floats
float calculateStdDev(const float* arr, int size, float mean) {
    float sum = 0.0f;
    for (int i = 0; i < size; ++i) {
        float diff = arr[i] - mean;
        sum += diff * diff;
    }
    return std::sqrt(sum / (size - 1));
}

__device__ float sumOfSquares3df(const float* vector) {
    return vector[0] * vector[0] + vector[1] * vector[1] + vector[2] * vector[2];
}
__device__ float sumOfSquares2df(const float* vector) {
    return vector[0] * vector[0] + vector[1] * vector[1];
}

__device__
void dist_noflip(float* point, RobotDimensions& dim, float* result_point)
// no angle flipping
{
    // Coxa as the frame of reference without rotation
    float result[3];
    result[0] = point[0] - dim.body;
    result[1] = point[1];
    result[2] = point[2];

    // finding coxa angle
    float required_angle_coxa = atan2f(result[1], result[0]);

    // flipping angle if above +-90deg
    // required_angle_coxa = fmodf(required_angle_coxa + dim.pI / 2.f + 2.f * dim.pI, dim.pI) - dim.pI / 2.f;

    // saturating coxa angle for dist
    required_angle_coxa = fmaxf(fminf(required_angle_coxa, dim.max_angle_coxa_w_margin), dim.min_angle_coxa_w_margin);

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox = cosf(-required_angle_coxa);
    float sin_angle_cox = sinf(-required_angle_coxa);
    float buffer = result[0] * sin_angle_cox;
    result[0] = result[0] * cos_angle_cox - result[1] * sin_angle_cox;
    result[1] = buffer + result[1] * cos_angle_cox;

    // Femur as the frame of reference witout rotation
    result[0] -= dim.coxa_length;

    // finding femur angle
    float required_angle_femur = atan2f(result[2], result[0]);

    // saturating coxa angle for dist
    required_angle_femur = fmaxf(fminf(required_angle_femur, dim.max_angle_femur_w_margin), dim.min_angle_femur_w_margin);

    // canceling femur rotation for dist
    float cos_angle_fem = cosf(required_angle_femur);
    float sin_angle_fem = sinf(required_angle_femur);

    // middle_TG as the frame of reference
    result[0] -= dim.middle_TG * cos_angle_fem;
    result[2] -= dim.middle_TG * sin_angle_fem;

    // rotating back to default xyz, but staying on middle_TG

    buffer = result[1] * sin_angle_cox;
    result[1] = -result[0] * sin_angle_cox + result[1] * cos_angle_cox;
    result[0] = result[0] * cos_angle_cox + buffer;

    result_point[0] = result[0];
    result_point[1] = result[1];
    result_point[2] = result[2];

    return;
}

__device__
void dist_flip(float* point, RobotDimensions& dim, float* result_point)
// with angle flipping
{
    // Coxa as the frame of reference without rotation
    float result[3];
    result[0] = point[0] - dim.body;
    result[1] = point[1];
    result[2] = point[2];

    // finding coxa angle
    float required_angle_coxa = atan2f(-result[1], -result[0]);

    // flipping angle if above +-90deg
    // required_angle_coxa = fmodf(required_angle_coxa + dim.pI / 2.f + 2.f * dim.pI, dim.pI) - dim.pI / 2.f;

    // saturating coxa angle for dist
    required_angle_coxa = fmaxf(fminf(required_angle_coxa, dim.max_angle_coxa_w_margin), dim.min_angle_coxa_w_margin);

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox = cosf(-required_angle_coxa);
    float sin_angle_cox = sinf(-required_angle_coxa);
    float buffer = result[0] * sin_angle_cox;
    result[0] = result[0] * cos_angle_cox - result[1] * sin_angle_cox;
    result[1] = buffer + result[1] * cos_angle_cox;

    // Femur as the frame of reference witout rotation
    result[0] -= dim.coxa_length;

    // finding femur angle
    float required_angle_femur = atan2f(result[2], result[0]);

    // saturating coxa angle for dist
    required_angle_femur = fmaxf(fminf(required_angle_femur, dim.max_angle_femur_w_margin), dim.min_angle_femur_w_margin);

    // canceling femur rotation for dist
    float cos_angle_fem = cosf(required_angle_femur);
    float sin_angle_fem = sinf(required_angle_femur);

    // middle_TG as the frame of reference
    result[0] -= dim.middle_TG * cos_angle_fem;
    result[2] -= dim.middle_TG * sin_angle_fem;

    // rotating back to default xyz, but staying on middle_TG

    buffer = result[1] * sin_angle_cox;
    result[1] = -result[0] * sin_angle_cox + result[1] * cos_angle_cox;
    result[0] = result[0] * cos_angle_cox + buffer;

    result_point[0] = result[0];
    result_point[1] = result[1];
    result_point[2] = result[2];

    return;
}

__device__
void dist_double_sol(float* point, RobotDimensions& dim, float* result_point)
//
{
    // Coxa as the frame of reference without rotation
    float result_noflip[3];
    float result_flip[3];

    dist_noflip(point, dim, result_noflip);
    dist_flip(point, dim, result_flip);

    float* result_to_use = (sumOfSquares3df(result_noflip) < sumOfSquares3df(result_flip)) ? result_noflip : result_flip;
    // result_to_use = result_flip;
    result_point[0] = result_to_use[0];
    result_point[1] = result_to_use[1];
    result_point[2] = result_to_use[2];

}

__device__
bool reachability(float* point, RobotDimensions& dim)
// no angle flipping
{
    // Coxa as the frame of reference without rotation
    float result[3];
    result[0] = point[0] - dim.body;
    result[1] = point[1];
    result[2] = point[2];

    // finding coxa angle
    float required_angle_coxa = atan2f(result[1], result[0]);

    // flipping angle if above +-90deg
    required_angle_coxa = fmodf(required_angle_coxa + dim.pI / 2.f + 2.f * dim.pI, dim.pI) - dim.pI / 2.f;

    if ((required_angle_coxa > dim.max_angle_coxa) || (required_angle_coxa < dim.min_angle_coxa)){
        return false;
    }

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox = cosf(-required_angle_coxa);
    float sin_angle_cox = sinf(-required_angle_coxa);
    float buffer = result[0] * sin_angle_cox;
    result[0] = result[0] * cos_angle_cox - result[1] * sin_angle_cox;
    result[1] = buffer + result[1] * cos_angle_cox;

    // Femur as the frame of reference witout rotation
    result[0] -= dim.coxa_length;

    float linnorm = norm3df(result[0], result[1], result[2]);

    if ((linnorm < dim.min_tibia_to_gripper_dist) || (linnorm > dim.max_tibia_to_gripper_dist)){
        return false;
    }

    // finding femur angle
    float required_angle_femur = atan2f(result[2], result[0]);

    if ((required_angle_femur > dim.min_angle_femur) && (required_angle_femur < dim.max_angle_femur)) {
        return true;
    }

    linnorm = fminf(
            norm3df(result[0] - dim.positiv_saturated_femur[0], 0, result[2] - dim.positiv_saturated_femur[1])
            ,
            norm3df(result[0] - dim.negativ_saturated_femur[0], 0, result[2] - dim.negativ_saturated_femur[1])
            );

    return linnorm < dim.femur_length;
}

__constant__ __device__ float data[][3] =
        {
                { 0.267004, 0.004874, 0.329415 },
                { 0.268510, 0.009605, 0.335427 },
                { 0.269944, 0.014625, 0.341379 },
                { 0.271305, 0.019942, 0.347269 },
                { 0.272594, 0.025563, 0.353093 },
                { 0.273809, 0.031497, 0.358853 },
                { 0.274952, 0.037752, 0.364543 },
                { 0.276022, 0.044167, 0.370164 },
                { 0.277018, 0.050344, 0.375715 },
                { 0.277941, 0.056324, 0.381191 },
                { 0.278791, 0.062145, 0.386592 },
                { 0.279566, 0.067836, 0.391917 },
                { 0.280267, 0.073417, 0.397163 },
                { 0.280894, 0.078907, 0.402329 },
                { 0.281446, 0.084320, 0.407414 },
                { 0.281924, 0.089666, 0.412415 },
                { 0.282327, 0.094955, 0.417331 },
                { 0.282656, 0.100196, 0.422160 },
                { 0.282910, 0.105393, 0.426902 },
                { 0.283091, 0.110553, 0.431554 },
                { 0.283197, 0.115680, 0.436115 },
                { 0.283229, 0.120777, 0.440584 },
                { 0.283187, 0.125848, 0.444960 },
                { 0.283072, 0.130895, 0.449241 },
                { 0.282884, 0.135920, 0.453427 },
                { 0.282623, 0.140926, 0.457517 },
                { 0.282290, 0.145912, 0.461510 },
                { 0.281887, 0.150881, 0.465405 },
                { 0.281412, 0.155834, 0.469201 },
                { 0.280868, 0.160771, 0.472899 },
                { 0.280255, 0.165693, 0.476498 },
                { 0.279574, 0.170599, 0.479997 },
                { 0.278826, 0.175490, 0.483397 },
                { 0.278012, 0.180367, 0.486697 },
                { 0.277134, 0.185228, 0.489898 },
                { 0.276194, 0.190074, 0.493001 },
                { 0.275191, 0.194905, 0.496005 },
                { 0.274128, 0.199721, 0.498911 },
                { 0.273006, 0.204520, 0.501721 },
                { 0.271828, 0.209303, 0.504434 },
                { 0.270595, 0.214069, 0.507052 },
                { 0.269308, 0.218818, 0.509577 },
                { 0.267968, 0.223549, 0.512008 },
                { 0.266580, 0.228262, 0.514349 },
                { 0.265145, 0.232956, 0.516599 },
                { 0.263663, 0.237631, 0.518762 },
                { 0.262138, 0.242286, 0.520837 },
                { 0.260571, 0.246922, 0.522828 },
                { 0.258965, 0.251537, 0.524736 },
                { 0.257322, 0.256130, 0.526563 },
                { 0.255645, 0.260703, 0.528312 },
                { 0.253935, 0.265254, 0.529983 },
                { 0.252194, 0.269783, 0.531579 },
                { 0.250425, 0.274290, 0.533103 },
                { 0.248629, 0.278775, 0.534556 },
                { 0.246811, 0.283237, 0.535941 },
                { 0.244972, 0.287675, 0.537260 },
                { 0.243113, 0.292092, 0.538516 },
                { 0.241237, 0.296485, 0.539709 },
                { 0.239346, 0.300855, 0.540844 },
                { 0.237441, 0.305202, 0.541921 },
                { 0.235526, 0.309527, 0.542944 },
                { 0.233603, 0.313828, 0.543914 },
                { 0.231674, 0.318106, 0.544834 },
                { 0.229739, 0.322361, 0.545706 },
                { 0.227802, 0.326594, 0.546532 },
                { 0.225863, 0.330805, 0.547314 },
                { 0.223925, 0.334994, 0.548053 },
                { 0.221989, 0.339161, 0.548752 },
                { 0.220057, 0.343307, 0.549413 },
                { 0.218130, 0.347432, 0.550038 },
                { 0.216210, 0.351535, 0.550627 },
                { 0.214298, 0.355619, 0.551184 },
                { 0.212395, 0.359683, 0.551710 },
                { 0.210503, 0.363727, 0.552206 },
                { 0.208623, 0.367752, 0.552675 },
                { 0.206756, 0.371758, 0.553117 },
                { 0.204903, 0.375746, 0.553533 },
                { 0.203063, 0.379716, 0.553925 },
                { 0.201239, 0.383670, 0.554294 },
                { 0.199430, 0.387607, 0.554642 },
                { 0.197636, 0.391528, 0.554969 },
                { 0.195860, 0.395433, 0.555276 },
                { 0.194100, 0.399323, 0.555565 },
                { 0.192357, 0.403199, 0.555836 },
                { 0.190631, 0.407061, 0.556089 },
                { 0.188923, 0.410910, 0.556326 },
                { 0.187231, 0.414746, 0.556547 },
                { 0.185556, 0.418570, 0.556753 },
                { 0.183898, 0.422383, 0.556944 },
                { 0.182256, 0.426184, 0.557120 },
                { 0.180629, 0.429975, 0.557282 },
                { 0.179019, 0.433756, 0.557430 },
                { 0.177423, 0.437527, 0.557565 },
                { 0.175841, 0.441290, 0.557685 },
                { 0.174274, 0.445044, 0.557792 },
                { 0.172719, 0.448791, 0.557885 },
                { 0.171176, 0.452530, 0.557965 },
                { 0.169646, 0.456262, 0.558030 },
                { 0.168126, 0.459988, 0.558082 },
                { 0.166617, 0.463708, 0.558119 },
                { 0.165117, 0.467423, 0.558141 },
                { 0.163625, 0.471133, 0.558148 },
                { 0.162142, 0.474838, 0.558140 },
                { 0.160665, 0.478540, 0.558115 },
                { 0.159194, 0.482237, 0.558073 },
                { 0.157729, 0.485932, 0.558013 },
                { 0.156270, 0.489624, 0.557936 },
                { 0.154815, 0.493313, 0.557840 },
                { 0.153364, 0.497000, 0.557724 },
                { 0.151918, 0.500685, 0.557587 },
                { 0.150476, 0.504369, 0.557430 },
                { 0.149039, 0.508051, 0.557250 },
                { 0.147607, 0.511733, 0.557049 },
                { 0.146180, 0.515413, 0.556823 },
                { 0.144759, 0.519093, 0.556572 },
                { 0.143343, 0.522773, 0.556295 },
                { 0.141935, 0.526453, 0.555991 },
                { 0.140536, 0.530132, 0.555659 },
                { 0.139147, 0.533812, 0.555298 },
                { 0.137770, 0.537492, 0.554906 },
                { 0.136408, 0.541173, 0.554483 },
                { 0.135066, 0.544853, 0.554029 },
                { 0.133743, 0.548535, 0.553541 },
                { 0.132444, 0.552216, 0.553018 },
                { 0.131172, 0.555899, 0.552459 },
                { 0.129933, 0.559582, 0.551864 },
                { 0.128729, 0.563265, 0.551229 },
                { 0.127568, 0.566949, 0.550556 },
                { 0.126453, 0.570633, 0.549841 },
                { 0.125394, 0.574318, 0.549086 },
                { 0.124395, 0.578002, 0.548287 },
                { 0.123463, 0.581687, 0.547445 },
                { 0.122606, 0.585371, 0.546557 },
                { 0.121831, 0.589055, 0.545623 },
                { 0.121148, 0.592739, 0.544641 },
                { 0.120565, 0.596422, 0.543611 },
                { 0.120092, 0.600104, 0.542530 },
                { 0.119738, 0.603785, 0.541400 },
                { 0.119512, 0.607464, 0.540218 },
                { 0.119423, 0.611141, 0.538982 },
                { 0.119483, 0.614817, 0.537692 },
                { 0.119699, 0.618490, 0.536347 },
                { 0.120081, 0.622161, 0.534946 },
                { 0.120638, 0.625828, 0.533488 },
                { 0.121380, 0.629492, 0.531973 },
                { 0.122312, 0.633153, 0.530398 },
                { 0.123444, 0.636809, 0.528763 },
                { 0.124780, 0.640461, 0.527068 },
                { 0.126326, 0.644107, 0.525311 },
                { 0.128087, 0.647749, 0.523491 },
                { 0.130067, 0.651384, 0.521608 },
                { 0.132268, 0.655014, 0.519661 },
                { 0.134692, 0.658636, 0.517649 },
                { 0.137339, 0.662252, 0.515571 },
                { 0.140210, 0.665859, 0.513427 },
                { 0.143303, 0.669459, 0.511215 },
                { 0.146616, 0.673050, 0.508936 },
                { 0.150148, 0.676631, 0.506589 },
                { 0.153894, 0.680203, 0.504172 },
                { 0.157851, 0.683765, 0.501686 },
                { 0.162016, 0.687316, 0.499129 },
                { 0.166383, 0.690856, 0.496502 },
                { 0.170948, 0.694384, 0.493803 },
                { 0.175707, 0.697900, 0.491033 },
                { 0.180653, 0.701402, 0.488189 },
                { 0.185783, 0.704891, 0.485273 },
                { 0.191090, 0.708366, 0.482284 },
                { 0.196571, 0.711827, 0.479221 },
                { 0.202219, 0.715272, 0.476084 },
                { 0.208030, 0.718701, 0.472873 },
                { 0.214000, 0.722114, 0.469588 },
                { 0.220124, 0.725509, 0.466226 },
                { 0.226397, 0.728888, 0.462789 },
                { 0.232815, 0.732247, 0.459277 },
                { 0.239374, 0.735588, 0.455688 },
                { 0.246070, 0.738910, 0.452024 },
                { 0.252899, 0.742211, 0.448284 },
                { 0.259857, 0.745492, 0.444467 },
                { 0.266941, 0.748751, 0.440573 },
                { 0.274149, 0.751988, 0.436601 },
                { 0.281477, 0.755203, 0.432552 },
                { 0.288921, 0.758394, 0.428426 },
                { 0.296479, 0.761561, 0.424223 },
                { 0.304148, 0.764704, 0.419943 },
                { 0.311925, 0.767822, 0.415586 },
                { 0.319809, 0.770914, 0.411152 },
                { 0.327796, 0.773980, 0.406640 },
                { 0.335885, 0.777018, 0.402049 },
                { 0.344074, 0.780029, 0.397381 },
                { 0.352360, 0.783011, 0.392636 },
                { 0.360741, 0.785964, 0.387814 },
                { 0.369214, 0.788888, 0.382914 },
                { 0.377779, 0.791781, 0.377939 },
                { 0.386433, 0.794644, 0.372886 },
                { 0.395174, 0.797475, 0.367757 },
                { 0.404001, 0.800275, 0.362552 },
                { 0.412913, 0.803041, 0.357269 },
                { 0.421908, 0.805774, 0.351910 },
                { 0.430983, 0.808473, 0.346476 },
                { 0.440137, 0.811138, 0.340967 },
                { 0.449368, 0.813768, 0.335384 },
                { 0.458674, 0.816363, 0.329727 },
                { 0.468053, 0.818921, 0.323998 },
                { 0.477504, 0.821444, 0.318195 },
                { 0.487026, 0.823929, 0.312321 },
                { 0.496615, 0.826376, 0.306377 },
                { 0.506271, 0.828786, 0.300362 },
                { 0.515992, 0.831158, 0.294279 },
                { 0.525776, 0.833491, 0.288127 },
                { 0.535621, 0.835785, 0.281908 },
                { 0.545524, 0.838039, 0.275626 },
                { 0.555484, 0.840254, 0.269281 },
                { 0.565498, 0.842430, 0.262877 },
                { 0.575563, 0.844566, 0.256415 },
                { 0.585678, 0.846661, 0.249897 },
                { 0.595839, 0.848717, 0.243329 },
                { 0.606045, 0.850733, 0.236712 },
                { 0.616293, 0.852709, 0.230052 },
                { 0.626579, 0.854645, 0.223353 },
                { 0.636902, 0.856542, 0.216620 },
                { 0.647257, 0.858400, 0.209861 },
                { 0.657642, 0.860219, 0.203082 },
                { 0.668054, 0.861999, 0.196293 },
                { 0.678489, 0.863742, 0.189503 },
                { 0.688944, 0.865448, 0.182725 },
                { 0.699415, 0.867117, 0.175971 },
                { 0.709898, 0.868751, 0.169257 },
                { 0.720391, 0.870350, 0.162603 },
                { 0.730889, 0.871916, 0.156029 },
                { 0.741388, 0.873449, 0.149561 },
                { 0.751884, 0.874951, 0.143228 },
                { 0.762373, 0.876424, 0.137064 },
                { 0.772852, 0.877868, 0.131109 },
                { 0.783315, 0.879285, 0.125405 },
                { 0.793760, 0.880678, 0.120005 },
                { 0.804182, 0.882046, 0.114965 },
                { 0.814576, 0.883393, 0.110347 },
                { 0.824940, 0.884720, 0.106217 },
                { 0.835270, 0.886029, 0.102646 },
                { 0.845561, 0.887322, 0.099702 },
                { 0.855810, 0.888601, 0.097452 },
                { 0.866013, 0.889868, 0.095953 },
                { 0.876168, 0.891125, 0.095250 },
                { 0.886271, 0.892374, 0.095374 },
                { 0.896320, 0.893616, 0.096335 },
                { 0.906311, 0.894855, 0.098125 },
                { 0.916242, 0.896091, 0.100717 },
                { 0.926106, 0.897330, 0.104071 },
                { 0.935904, 0.898570, 0.108131 },
                { 0.945636, 0.899815, 0.112838 },
                { 0.955300, 0.901065, 0.118128 },
                { 0.964894, 0.902323, 0.123941 },
                { 0.974417, 0.903590, 0.130215 },
                { 0.983868, 0.904867, 0.136897 },
                { 0.993248, 0.906157, 0.143936 }
        };

__device__ void linearInterp(float* x, float* result, float** colorMap)
{
    const float x_clamped  = fmaxf(fminf(255.f, *x), 0.f);
    const float x_stepped  = floorf(x_clamped);
    const float t  = x_stepped - *x;
    const float* c1  = colorMap[static_cast<int>(x_stepped)];
    const float* c2  = colorMap[static_cast<int>(x_stepped)+1];

    result[0] = c1[0]*(1-t) + c2[0]*(t);
    result[1] = c1[1]*(1-t) + c2[1]*(t);
    result[2] = c1[2]*(1-t) + c2[2]*(t);
    result[3] = 1;
}

__global__ void toVirdisUint_kernel(Matrixf table, unsigned char* pixels) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < table.height; i += stride) {
        const int x = (int) floorf(
                fmaxf(
                        fminf(255.f, table.elements[i]),
                        0.f)
        );
        const float color[3] = {data[x][0], data[x][1], data[x][2]};
        pixels[i * 4 + 0] = (unsigned char) floorf(color[0]*255.f);
        pixels[i * 4 + 1] = (unsigned char) floorf(color[1]*255.f);
        pixels[i * 4 + 2] = (unsigned char) floorf(color[2]*255.f);
        pixels[i * 4 + 3] = (unsigned char) (1*255);
    }
}

// Kernel launch function
__global__
void dist_kernel(Matrixf table, RobotDimensions dimensions, Matrixf result_table)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        dist_double_sol((table.elements + i * table.width), dimensions, (result_table.elements + i * result_table.width));
    }
}

__global__
void switch_zy_kernel(Matrixf table)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        float buffer = table.elements[i * table.width + 1];
        table.elements[i * table.width + 1] = table.elements[i * table.width + 2];
        table.elements[i * table.width + 2] = buffer;
    }
}

__global__
void norm3df_kernel(Matrixf table, Matrixf result_table)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        result_table.elements[i] = norm3df(table.elements[i * table.width],
                                           table.elements[i * table.width+1],
                                           table.elements[i * table.width+2]);
    }
}

__global__
void change_z_kernel(Matrixf table, float zval)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        table.elements[i * table.width+2] = zval;
    }
}

__global__
void change_y_kernel(Matrixf table, float zval)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        table.elements[i * table.width+1] = zval;
    }
}

__global__
void dist2virdis_pipeline(Matrixf table, RobotDimensions dimensions, unsigned char* pixels)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        float result[3];
        // dist func
        dist_double_sol((table.elements + i * table.width), dimensions, result);

        //norm func
        // norm stored in result[0]
        result[0] = norm3df(result[0],
                                           result[1],
                                           result[2])
                                                   /2;

        // virdis colormaping

        const int x = (int)
                floorf(
                fmaxf(
                        fminf(255.f, result[0]),
                        0.f)
        );
        const float color[3] = {data[x][0], data[x][1], data[x][2]};
        pixels[i * 4 + 0] = (unsigned char) floorf(color[0]*255.f);
        pixels[i * 4 + 1] = (unsigned char) floorf(color[1]*255.f);
        pixels[i * 4 + 2] = (unsigned char) floorf(color[2]*255.f);
        pixels[i * 4 + 3] = (unsigned char) (1*255);
    }
}

__global__
void reachability2img_pipeline(Matrixf table, RobotDimensions dimensions, unsigned char* pixels)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < table.height; i += stride) {
        bool result = reachability((table.elements + i * table.width), dimensions);

        unsigned char val = (result)? 0 : 255;
        for (int n=0; n<4; n++){
            pixels[i * 4 + n] = val;
        }
        pixels[i * 4 + 3] = (unsigned char) (1*255);
    }
}

__device__
bool legs_reachable(float* body_position, Matrixf target_set, RobotDimensions dimensions){
//    return true;
    float point_relativ_to_body[3];
    bool result = true;
    float sin_buffer;
    float rot_angle;
    float* leg_target;

    for (int leg = 0; ((leg < 6) && result); leg ++) {
        leg_target = (target_set.elements + leg * 3);

        point_relativ_to_body[0] = leg_target[0] - body_position[0];
        point_relativ_to_body[1] = leg_target[1] - body_position[1];
        point_relativ_to_body[2] = leg_target[2] - body_position[2];

        rot_angle = -dimensions.pI/3.f * (float)leg;
        sin_buffer = sin(rot_angle) * point_relativ_to_body[0];

        point_relativ_to_body[0] =
                cos(rot_angle) * point_relativ_to_body[0] -
                sin(rot_angle) * point_relativ_to_body[1];
        point_relativ_to_body[1] =
                sin_buffer +
                cos(rot_angle) * point_relativ_to_body[1];

        result = reachability(point_relativ_to_body, dimensions);
    }
    return result;
}

__global__
void all5_reachable(Matrixf body_pos_table, RobotDimensions dimensions, Matrixf target_set, unsigned char* pixels)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < body_pos_table.height; i += stride) {

        float* body_pos = (body_pos_table.elements + i * body_pos_table.width);
        float point_relativ_to_body[3];
        bool result = true;

        for (int leg = 1; ((leg < 6) && result); leg ++) {
            float* leg_target = (target_set.elements + leg * 3);

            point_relativ_to_body[0] = leg_target[0] - body_pos[0];
            point_relativ_to_body[1] = leg_target[1] - body_pos[1];
            point_relativ_to_body[2] = leg_target[2] - body_pos[2];

            float rot_angle = -dimensions.pI/3.f * (float)leg;
            float sin_buffer = sin(rot_angle) * point_relativ_to_body[0];
            point_relativ_to_body[0] =
                    cos(rot_angle) * point_relativ_to_body[0] -
                    sin(rot_angle) * point_relativ_to_body[1];
            point_relativ_to_body[1] =
                    sin_buffer +
                    cos(rot_angle) * point_relativ_to_body[1];

            result = reachability(point_relativ_to_body, dimensions);
        }

        unsigned char val = (result)? 0 : 255;
        for (int n=0; n<4; n++){
            pixels[i * 4 + n] = val;
        }
        pixels[i * 4 + 3] = (unsigned char) (1*255);
    }
}

__global__
void accumulate_leg0_movable(Matrixf body_pos_table,
                             Matrixf output_pos_table,
                             Matrixf target_set,
                             int* accumulator,
                             RobotDimensions dimensions)
{
    int index = (int)blockIdx.x * blockDim.x + threadIdx.x;
    int stride = (int)blockDim.x * gridDim.x;
    float* leg_target;
    float* body_pos;
    Matrixf local_target_set;
    local_target_set.width = 3; local_target_set.height = 6;
    float test[18];
    local_target_set.elements = test;
    for (int i = 0; i < 18; i ++){
        local_target_set.elements[i] = target_set.elements[i];
    }

    for (int i = index; i < (body_pos_table.height * output_pos_table.height); i += stride) {
        int body_index = i % (body_pos_table.height);
        int out_index = i / output_pos_table.height;
        atomicExch(&accumulator[out_index], 255);

//        leg_target = output_pos_table.elements + out_index * 3;
//        body_pos = body_pos_table.elements + body_index * 3;
//
//        local_target_set.elements[0] = leg_target[0];
//        local_target_set.elements[1] = leg_target[1];
//        local_target_set.elements[2] = 0;
//
//        bool result = legs_reachable(body_pos, local_target_set, dimensions);
//
//        if (result){
//            atomicAdd(&accumulator[out_index], 1);
//        }
    }
}

__global__
void accumulate_leg0_movablev2(Matrixf body_pos_table,
                             Matrixf output_pos_table,
                             Matrixf target_set,
                             int* accumulator,
                             RobotDimensions dimensions)
{
    int indexA = blockIdx.x * blockDim.x + threadIdx.x;
    int indexB = blockIdx.y * blockDim.y + threadIdx.y;
    int strideA = blockDim.x * gridDim.x;
    int strideB = blockDim.y * gridDim.y;
    float* leg_target;
    float* body_pos;
    Matrixf local_target_set;
    local_target_set.width = 3; local_target_set.height = 6;
    float test[18];
    local_target_set.elements = test;
    for (int i = 0; i < 18; i ++){
        local_target_set.elements[i] = target_set.elements[i];
    }

    for (int a = indexA; a < (body_pos_table.height); a += strideA) {
        for (int b = indexB; b < (output_pos_table.height); b += strideB) {

            leg_target = output_pos_table.elements + b * 3;
            body_pos = body_pos_table.elements + a * 3;
            local_target_set.elements[0] = leg_target[0];
            local_target_set.elements[1] = leg_target[1];
            local_target_set.elements[2] = 0;
            bool result = legs_reachable(body_pos, local_target_set, dimensions);

            if (result){
//                atomicAdd(&accumulator[out_index], 1);
                atomicAdd(&accumulator[b], 1);
            }


        }
    }
}

__global__
void accumulator_to_pixel(int* accumulator, unsigned char* pixels, int number_of_pixels)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_pixels; i += stride) {
        auto val = (unsigned char) min(max(accumulator[i], 0), 255);
//        auto val = (unsigned char) min(max(i, 0), 255);

        for (int n=0; n<4; n++){
            pixels[i * 4 + n] = val;
        }
        pixels[i * 4 + 3] = (unsigned char) (255);
    }

}

AutoEstimator::AutoEstimator(int pxWidth, int pxHeight) {
    blockSize = 1024;
    verbose = true;
    screenWidth = pxWidth;
    screenHeight = pxHeight;
    dimensions = dim_of_SCARE();
    rows = screenWidth * screenHeight;

    table_input.width = 3; table_input.height = rows;
    table_input.elements = new float[table_input.width * table_input.height];

    result.width = 3; result.height = rows;
    result.elements = new float[result.width * result.height];

    result_norm.width = 1; result_norm.height = rows;
    result_norm.elements = new float[result_norm.width * result_norm.height];

    virdisTexture = new unsigned char [4 * rows];

    targetset.width = 3; targetset.height = 6;
    targetset.elements = new float[targetset.width * targetset.height];

    for (int leg=0; leg < 6; leg++) {
        targetset.elements[leg*3 + 0] = cos(3.14159f/3.f * (float)leg) * 420;
        targetset.elements[leg*3 + 1] = sin(3.14159f/3.f * (float)leg) * 420;
        targetset.elements[leg*3 + 2] = 0.f;
    }

    table_input_gpu.width = table_input.width; table_input_gpu.height = table_input.height;
    result_gpu.width = table_input_gpu.width; result_gpu.height = table_input_gpu.height;
    result_norm_gpu.width = result_norm.width; result_norm_gpu.height = table_input_gpu.height;
    targetset_gpu.width = 3; targetset_gpu.height = 6;

    numBlocks = (rows + blockSize - 1) / blockSize;
    error_check();
    setup_kernel();
    input_as_grid();
    allocate_gpu_mem();
    copy_input_cpu2gpu();
}

void AutoEstimator::error_check(){
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda error: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error or exit the program as needed.
    }
}

void AutoEstimator::input_as_grid(){
    for (int i = 0; i < screenHeight; i++) {
        for (int j = 0; j < screenWidth; j++) {
            int row = i * screenWidth + j;
            // X
            *(table_input.elements + row * table_input.width + 0)
                    //= -1000.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 2000.0f;
                    = (float)(j - screenWidth /2)*20;
            // Y
            *(table_input.elements + row * table_input.width + 1)
                    //= -1000.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 2000.0f;
                    = -(float)(i - screenHeight / 2)*20;
            // Z
            *(table_input.elements + row * table_input.width + 2)
                    //= -500.0f + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 1000.0f;
                    = 0.f;
        }
    }
    if (verbose) { std::cout << "Host grid generated" << std::endl; }
}

void AutoEstimator::change_z_value(float value){

    change_z_kernel<<<numBlocks, blockSize >>>(table_input_gpu, value);
    hipDeviceSynchronize(); error_check();
}

void AutoEstimator::change_y_value(float value){

    change_y_kernel<<<numBlocks, blockSize >>>(table_input_gpu, value);
    hipDeviceSynchronize(); error_check();
}

void AutoEstimator::switch_zy(){

    switch_zy_kernel<<<numBlocks, blockSize >>>(table_input_gpu);
    hipDeviceSynchronize(); error_check();
}

void AutoEstimator::allocate_gpu_mem(){
    hipMalloc(&table_input_gpu.elements, table_input_gpu.width * table_input_gpu.height * sizeof(float));

    hipMalloc(&result_gpu.elements, result_gpu.width * result_gpu.height * sizeof(float));
    hipMemset(result_gpu.elements, 0, rows * sizeof(int));

    hipMalloc(&result_norm_gpu.elements, result_norm_gpu.width * result_norm_gpu.height * sizeof(float));

    hipMalloc(&virdisTexture_gpu, 4 * rows * sizeof(unsigned char));

    hipMalloc(&targetset_gpu.elements, targetset_gpu.width * targetset_gpu.height * sizeof(float));

    hipMalloc(&gpu_accumulator, rows * sizeof(int));
    hipMemset(gpu_accumulator, 0, rows * sizeof(int));

    if (verbose) { std::cout << "GPU memory allocated" << std::endl; }
    error_check();
}

void AutoEstimator::copy_input_cpu2gpu(){
    hipMemcpy(table_input_gpu.elements,
               table_input.elements,
               table_input.width * table_input.height * sizeof(float),
               hipMemcpyHostToDevice);

    hipMemcpy(targetset_gpu.elements,
               targetset.elements,
               targetset.width * targetset.height * sizeof(float),
               hipMemcpyHostToDevice);

    if (verbose) { std::cout << "Host data copied to GPU" << std::endl; }
    error_check();
}

void AutoEstimator::setup_kernel(){
    std::wcout << "Threads per block: " << blockSize << "\nNumber of blocks: " << numBlocks << std::endl;
    empty_kernel<<<numBlocks, blockSize >>>();
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Empty kernels started" << std::endl;}
    error_check();
}

void AutoEstimator::compute_dist(){
    if (verbose) { std::cout << "Compute started" << std::endl;}
    dist_kernel<<<numBlocks, blockSize >>>(table_input_gpu, dimensions,
                                           result_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::compute_result_norm(){
    if (verbose) { std::cout << "Compute norm started" << std::endl;}
    norm3df_kernel<<<numBlocks, blockSize >>>(result_gpu,
                                              result_norm_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::convert_to_virdis(){
    if (verbose) { std::cout << "Compute virdis started" << std::endl;}
    toVirdisUint_kernel<<<numBlocks, blockSize >>>(result_norm_gpu,
                                                   virdisTexture_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::dist_to_virdis_pipeline(){
    if (verbose) { std::cout << "dist_to_virdis_pipeline started" << std::endl;}
    dist2virdis_pipeline<<<numBlocks, blockSize >>>(table_input_gpu, dimensions,
                                                   virdisTexture_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::reachability_to_img_pipeline(){
    if (verbose) { std::cout << "dist_to_virdis_pipeline started" << std::endl;}
    reachability2img_pipeline<<<numBlocks, blockSize >>>(table_input_gpu, dimensions,
                                                    virdisTexture_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::all_reachable_default_to_image(){
    if (verbose) { std::cout << "dist_to_virdis_pipeline started" << std::endl;}
    all5_reachable<<<numBlocks, blockSize >>>(table_input_gpu,
                                              dimensions,
                                              targetset_gpu,
                                              virdisTexture_gpu);
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
}

void AutoEstimator::compute_leg0_by_accumulation(){
    if (verbose) { std::cout << "dist_to_virdis_pipeline started" << std::endl;}
    // (rows*rows + blockSize - 1) / blockSize
    dim3 blockDim2d(16, 16);
    dim3 gridDim2d((rows + blockDim2d.x - 1) / blockDim2d.x, (rows + blockDim2d.y - 1) / blockDim2d.y);
    accumulate_leg0_movablev2<<<gridDim2d, blockDim2d >>>(
            table_input_gpu,
            table_input_gpu,
            targetset_gpu,
            gpu_accumulator,
            dimensions);
    hipDeviceSynchronize();
    error_check();
//    throw std::runtime_error("stopped");
    accumulator_to_pixel<<<numBlocks, blockSize >>>(
            gpu_accumulator,
            virdisTexture_gpu,
            rows);
    hipDeviceSynchronize();
    hipMemset(gpu_accumulator, 0, rows * sizeof(int));
    hipDeviceSynchronize();
    if (verbose) { std::cout << "Compute done" << std::endl;}
    error_check();
//    throw std::runtime_error("stopped");
}

void AutoEstimator::copy_output_gpu2cpu(){
    hipMemcpy(result.elements,
               result_gpu.elements,
               result_gpu.width * result_gpu.height * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(virdisTexture,
               virdisTexture_gpu,
               4 * rows * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    if (verbose) { std::cout << "GPU result copied to host" << std::endl;}
    hipDeviceSynchronize();
    error_check();
}

void AutoEstimator::virdisresult_gpu2cpu(){
    hipMemcpy(virdisTexture,
               virdisTexture_gpu,
               4 * rows * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    if (verbose) { std::cout << "virdis result copied to host" << std::endl;}
    hipDeviceSynchronize();
    error_check();
}

void AutoEstimator::delete_all(){
    delete[] table_input.elements;
    hipFree(table_input_gpu.elements);
    hipFree(result_gpu.elements);
    hipFree(result_norm_gpu.elements);
    hipFree(virdisTexture_gpu);
    delete[] result.elements;
    delete[] result_norm.elements;
    delete[] virdisTexture;
    if (verbose) { std::cout << "All pointers deleted" << std::endl;}
    error_check();
}