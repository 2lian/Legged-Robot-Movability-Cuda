#include "hip/hip_runtime.h"
#include "HeaderCPP.h"
#include "HeaderCUDA.h"
#include "collision.cu.h"
#include "one_leg.cu.h"
#include "thrust/detail/copy.h"
#include "thrust/detail/raw_pointer_cast.h"
#include "thrust/device_vector.h"
#include "thrust/fill.h"
#include "thrust/transform.h"
#include <cstdio>
#include <iostream>

__device__ void rotateInPlace(float3& point, float z_rot, float& cos_memory,
                              float& sin_memory) {
    sincosf(z_rot, &sin_memory, &cos_memory);
    float buffer = point.x * sin_memory;
    point.x = point.x * cos_memory - point.y * sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

__device__ void unrotateInPlace(float3& point, float z_rot, float& cos_memory,
                                float& sin_memory) {
    float buffer = point.x * -sin_memory;
    point.x = point.x * cos_memory - point.y * -sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

__device__ bool reachable_rotate_leg(float3 target, const float3 body_pos,
                                     const LegDimensions& dim) {
    {
        float cos_memory;
        float sin_memory;
        target.x -= body_pos.x;
        target.y -= body_pos.y;
        target.z -= body_pos.z;
        rotateInPlace(target, -dim.body_angle, cos_memory, sin_memory);
    }
    return reachability_absolute_tibia_limit(target, dim);
};

__global__ void reachable_leg_kernel_accu(Array<float3> body_map,
                                          Array<float3> target_map,
                                          LegDimensions dim,
                                          Array<int> output) {
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    long stride = blockDim.x * gridDim.x;
    long maxid = (long)body_map.length * (long)target_map.length;
    for (long i = index; i < maxid; i += stride) {
        int body_index = i / target_map.length;
        int target_index = i % target_map.length;
        // if (output.elements[body_index] == -1) {
        //     return;
        // }
        float3& target = target_map.elements[target_index];
        float3& body_pos = body_map.elements[body_index];
        if (reachable_rotate_leg(target, body_pos, dim) or true) {
            atomicAdd(&output.elements[body_index], 1);
        }
    }
};

__global__ void find_min_kernel(Array<int>* arrays, int number_of_legs,
                                Array<int> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < output.length; i += stride) {
        // for (int i = index; i < 1; i += stride) {
        int min_value = arrays[0].elements[i];

        for (int leg = 1; leg < number_of_legs; ++leg) {
            min_value = min(min_value, arrays[leg].elements[i]);
            // min_value = min_value + arrays[leg].elements[i];
        }

        output.elements[i] = min_value;
    }
}

__global__ void find_min_kernel(thrust::device_vector<int>* arrays,
                                int number_of_legs,
                                thrust::device_vector<int> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < output.size(); i += stride) {
        // for (int i = index; i < 1; i += stride) {
        int min_value = arrays[0][i];

        for (int leg = 1; leg < number_of_legs; ++leg) {
            // min_value = min(min_value, arrays[leg][i]);
            min_value = min_value + arrays[leg][i];
        }
        output[i] = min_value;
    }
}

#define CUDA_CHECK_ERROR(errorMessage)                                         \
    do {                                                                       \
        hipError_t err = hipGetLastError();                                  \
        if (err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error in %s: %s\n", errorMessage,            \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

Array<int> robot_full_reachable(Array<float3> body_map,
                                Array<float3> target_map,
                                Array<LegDimensions> legs) {
    {
        float3* newpointer;
        hipMalloc(&newpointer, body_map.length * sizeof(float3));
        CUDA_CHECK_ERROR("hipMalloc body_map");
        hipMemcpy(newpointer, body_map.elements,
                   body_map.length * sizeof(float3), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR("hipMemcpy body_map");
        body_map.elements = newpointer;
    }
    {
        float3* newpointer;
        hipMalloc(&newpointer, target_map.length * sizeof(float3));
        CUDA_CHECK_ERROR("hipMalloc target_map");
        hipMemcpy(newpointer, target_map.elements,
                   target_map.length * sizeof(float3), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR("hipMemcpy target_map");
        target_map.elements = newpointer;
    }
    CUDA_CHECK_ERROR("hipMalloc before leg");

    Array<int>* res_bool_array;
    res_bool_array = new Array<int>[legs.length];
    int blockSize = 1024 / 1;
    int numBlock =
        (body_map.length * target_map.length + blockSize - 1) / blockSize;

    for (int leg_num = 0; leg_num < legs.length; leg_num++) {
        res_bool_array[leg_num].length = body_map.length;
        hipMalloc(&(res_bool_array[leg_num].elements),
                   sizeof(int) * res_bool_array[leg_num].length);
        if (leg_num == 0) {
            hipMemset(res_bool_array[leg_num].elements, 0,
                       sizeof(int) * res_bool_array[leg_num].length);
            float radius = legs.elements[0].body;
            float plus_z = 120;
            float minus_z = -60;
            in_cylinder_accu_kernel<<<numBlock, blockSize>>>(
                body_map, target_map, res_bool_array[leg_num], radius, plus_z,
                minus_z);
        } else {
            hipMemcpy(res_bool_array[leg_num].elements,
                       res_bool_array[0].elements,
                       sizeof(int) * res_bool_array[leg_num].length,
                       hipMemcpyDeviceToDevice);
        }
    }
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Cylinder and alloc");
    blockSize = 1024 / 2;
    numBlock =
        (body_map.length * target_map.length + blockSize - 1) / blockSize;
    for (int leg_num = 0; leg_num < legs.length; leg_num++) {
        CUDA_CHECK_ERROR("cudaKernel leg");
        reachable_leg_kernel_accu<<<numBlock, blockSize>>>(
            body_map, target_map, legs.elements[leg_num],
            res_bool_array[leg_num]);
    }
    hipDeviceSynchronize();

    CUDA_CHECK_ERROR("hipMalloc reachable kernel");

    Array<int> final_count;
    final_count.length = res_bool_array[0].length;

    hipMalloc(&final_count.elements, sizeof(int) * final_count.length);
    hipMemset(final_count.elements, 0, final_count.length * sizeof(int));
    CUDA_CHECK_ERROR("hipMalloc final_count");

    {
        Array<int>* newpointer;
        hipMalloc(&newpointer, legs.length * sizeof(Array<int>));
        CUDA_CHECK_ERROR("hipMalloc legdim");
        hipMemcpy(newpointer, res_bool_array, legs.length * sizeof(Array<int>),
                   hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR("hipMemcpy legdim");
        res_bool_array = newpointer;
    }

    blockSize = 1024;
    numBlock = (body_map.length + blockSize - 1) / blockSize;
    find_min_kernel<<<numBlock, blockSize>>>(res_bool_array, legs.length,
                                             final_count);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("execution find_min_kernel");
    int* newpointer = new int[final_count.length];
    {
        hipMemcpy(newpointer, final_count.elements,
                   final_count.length * sizeof(int), hipMemcpyDeviceToHost);
        CUDA_CHECK_ERROR("hipMemcpy final_count");
        hipFree(final_count.elements);
        final_count.elements = newpointer;
    }

    hipFree(body_map.elements);
    hipFree(target_map.elements);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("hipFree");
    return final_count;
}

struct CylinderFunctor {
    float radius;
    float plus_z;
    float minus_z;

    __host__ __device__ CylinderFunctor(float _radius, float _plus_z,
                                        float _minus_z)
        : radius(_radius), plus_z(_plus_z), minus_z(_minus_z) {}

    __device__ int operator()(float3 body, float3 target) const {
        bool r = in_cylinder(radius, plus_z, minus_z, body, target);
        // bool r = in_cylinder(10, 10, 10, body, target);
        return r ? -1 : 0;
    }
};

Array<int> robot_full_cccl(Array<float3> body_map, Array<float3> target_map,
                           Array<LegDimensions> legs) {
    thrust::host_vector<float3> Body(body_map.elements,
                                     body_map.elements + body_map.length);
    thrust::host_vector<float3> Target(target_map.elements,
                                       target_map.elements + target_map.length);

    thrust::device_vector<float3> Body_g = Body;
    thrust::device_vector<float3> Target_g = Target;

    thrust::device_vector<int>* result_ar;

    result_ar = new thrust::device_vector<int>[legs.length];
    int blockSize = 1024 / 1;
    int numBlock =
        (Body_g.size() * Target_g.size() + blockSize - 1) / blockSize;

    for (int leg_num = 0; leg_num < legs.length; leg_num++) {
        thrust::device_vector<int>& this_leg_result = result_ar[leg_num];
        this_leg_result.resize(Body_g.size());
        if (leg_num == 0) {
            thrust::fill(this_leg_result.begin(), this_leg_result.end(), 0);
            float radius = legs.elements[0].body;
            float plus_z = 120;
            float minus_z = -60;
            CylinderFunctor cylinder_func(radius, plus_z, minus_z);
            in_cylinder_cccl_kernel<<<numBlock, blockSize>>>(
                thrust::raw_pointer_cast(Body_g.data()), Body_g.size(),
                thrust::raw_pointer_cast(Target_g.data()), Target_g.size(),
                thrust::raw_pointer_cast(this_leg_result.data()), radius,
                plus_z, minus_z);
            hipDeviceSynchronize();

            thrust::device_vector<float3>::iterator newEndBody =
                thrust::remove_if(Body_g.begin(), Body_g.end(),
                                  this_leg_result.begin(),
                                  [] __device__(int x) { return x == -1; });
            Body_g.erase(newEndBody, Body_g.end());
            std::cout << Body_g.size() << std::endl;
            thrust::device_vector<int>::iterator newEndResult = thrust::remove(
                this_leg_result.begin(), this_leg_result.end(), -1);
            this_leg_result.erase(newEndResult, this_leg_result.end());
            std::cout << this_leg_result.size() << std::endl;

        } else {
            thrust::copy(result_ar[0].begin(), result_ar[0].end(),
                         this_leg_result.begin());
        }
    }
    CUDA_CHECK_ERROR("Cylinder and alloc");
    blockSize = 1024 / 2;
    numBlock = (Body_g.size() * Target_g.size() + blockSize - 1) / blockSize;
    for (int leg_num = 0; leg_num < legs.length; leg_num++) {
        CUDA_CHECK_ERROR("cudaKernel leg");
        Array<float3> b = {Body_g.size(),
                           thrust::raw_pointer_cast(Body_g.data())};
        Array<float3> t = {Target_g.size(),
                           thrust::raw_pointer_cast(Target_g.data())};
        Array<int> r = {result_ar[leg_num].size(),
                        thrust::raw_pointer_cast(result_ar[leg_num].data())};
        reachable_leg_kernel_accu<<<numBlock, blockSize>>>(
            b, t, legs.elements[leg_num], r);
    hipDeviceSynchronize();
        int sum = thrust::reduce(result_ar[leg_num].begin(),
                                 result_ar[leg_num].end(), 0);
        std::cout << "Sum of the elements: " << sum << std::endl;
    }
    hipDeviceSynchronize();

    CUDA_CHECK_ERROR("hipMalloc reachable kernel");

    thrust::device_vector<int> final_count(result_ar[0].size());

    CUDA_CHECK_ERROR("hipMalloc final_count");

    thrust::device_vector<int>* result_ar_gpu;
    hipMalloc(&result_ar_gpu,
               legs.length * sizeof(thrust::device_vector<int>));
    CUDA_CHECK_ERROR("hipMalloc legdim");
    hipMemcpy(result_ar_gpu, result_ar,
               legs.length * sizeof(thrust::device_vector<int>),
               hipMemcpyHostToDevice);
    // delete result_ar;
    CUDA_CHECK_ERROR("hipMemcpy legdim");
    result_ar_gpu = result_ar_gpu;

    blockSize = 1024;
    numBlock = (Body_g.size() + blockSize - 1) / blockSize;
    find_min_kernel<<<numBlock, blockSize>>>(result_ar_gpu, legs.length,
                                             final_count);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("execution find_min_kernel");

    thrust::device_vector<float3>::iterator newEndBody =
        thrust::remove_if(Body_g.begin(), Body_g.end(), final_count.begin(),
                          [] __device__(int x) { return x == 0; });
    Body_g.erase(newEndBody, Body_g.end());
    std::cout << Body_g.size() << std::endl;
    thrust::device_vector<int>::iterator newEndResult =
        thrust::remove(final_count.begin(), final_count.end(), 0);
    final_count.erase(newEndResult, final_count.end());
    std::cout << final_count.size() << std::endl;

    Array<int> out;
    thrust::host_vector<int> oncpu = final_count;
    out.length = oncpu.size();
    out.elements = new int[out.length];
    std::copy(oncpu.data(), oncpu.data() + oncpu.size(), out.elements);

    CUDA_CHECK_ERROR("hipFree");
    return out;
}
