#include "hip/hip_runtime.h"
#include "HeaderCPP.h"
#include "HeaderCUDA.h"
#include "collision.cu.h"
#include "hip/hip_runtime_api.h"
#include "cuda_util.cuh"
#include "one_leg.cu.h"
#include "thrust/copy.h"
#include "thrust/remove.h"
#include "unified_math_cuda.cu.h"
#include <algorithm>
#include <chrono>
#include <csignal>
#include <cstdio>
#include <iostream>
#include <iterator>
#include <limits>
#include <ostream>
#include <thrust/detail/copy.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/partition.h>
#include <thrust/transform.h>
#include <tuple>

__device__ inline void rotateInPlace(float3& point, float z_rot,
                                     float& cos_memory, float& sin_memory) {
    sincosf(z_rot, &sin_memory, &cos_memory);
    float buffer = point.x * sin_memory;
    point.x = point.x * cos_memory - point.y * sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

__device__ inline void unrotateInPlace(float3& point, float z_rot,
                                       float& cos_memory, float& sin_memory) {
    float buffer = point.x * -sin_memory;
    point.x = point.x * cos_memory - point.y * -sin_memory;
    point.y = buffer + point.y * cos_memory;
    return;
}

__device__ inline bool simple_grav_consideration(float3 target,
                                                 Quaternion orientation) {
    float3 gravity_down = qtRotate(qtInvert(orientation), target);
    return gravity_down.x < 0;
}

__device__ inline bool reachable_rotate_leg(float3 target,
                                            const float3 body_pos,
                                            const Quaternion orientation,
                                            const LegDimensions& dim) {
    {
        float cos_memory;
        float sin_memory;
        target.x -= body_pos.x;
        target.y -= body_pos.y;
        target.z -= body_pos.z;

        float3 gravity_down = qtRotate(qtInvert(orientation), target);
        rotateInPlace(gravity_down, -dim.body_angle, cos_memory, sin_memory);
        if (gravity_down.x < 0) {
            return false;
        }
        rotateInPlace(target, -dim.body_angle, cos_memory, sin_memory);
        // if (target.x < 0) {
        // return false;
        // }
    }
    // return false;
    return reachability_absolute_tibia_limit(target, dim);
};

__global__ void reachable_leg_kernel_accu(Array<float3> body_map,
                                          Array<float3> target_map,
                                          LegDimensions dim,
                                          Quaternion orientation,
                                          Array<int> output) {
    __shared__ LegDimensions sdim;
    if (threadIdx.x == 0) {
        sdim = dim;
    }
    __syncthreads();

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t maxid = (size_t)body_map.length * (size_t)target_map.length;
    for (size_t i = index; i < maxid; i += stride) {
        size_t body_index = i / target_map.length;
        size_t target_index = i % target_map.length;
        float3& target = target_map.elements[target_index];
        float3& body_pos = body_map.elements[body_index];
        if (reachable_rotate_leg(target, body_pos, orientation, sdim)) {
            atomicAdd(&output.elements[body_index], 1);
        }
    }
};

__global__ void reach_mem_kernel(float3* body_centers, const size_t Nb,
                                 float3* targets, const size_t Nt,
                                 unsigned char* output, const LegDimensions dim,
                                 const Quaternion orientation) {
    __shared__ LegDimensions sdim;
    __shared__ Quaternion sorientation;
    __shared__ bool one_reachable;
    __shared__ float3 body_pos;

    auto center_index = blockIdx.x;
    auto target_index = threadIdx.x;

    if (target_index == 0) {
        sdim = dim;
        sorientation = orientation;
        one_reachable = false;
        body_pos = body_centers[center_index];
    }
    __syncthreads();

    if ((center_index < Nb) and (target_index < Nt)) {
        const float3 target = targets[target_index];
        if (reachable_rotate_leg(target, body_pos, sorientation, sdim)) {
            one_reachable = true;
        }
    }

    __syncthreads();
    if ((target_index == 0) && (one_reachable)) {
        output[center_index] = 1;
    }
};

void launch_opti_mem_reach_kernel(Array<float3> body_map,
                                  Array<float3> target_map, LegDimensions dim,
                                  Quaternion orientation,
                                  Array<unsigned char> output) {
    size_t processed_index = 0;
    size_t max_block_size = 1024 / 2;

    auto Nc = body_map.length;
    auto body_centers = body_map.elements;
    auto Nt = target_map.length;
    auto targets = target_map.elements;

    size_t numBlock = Nc;
    while (processed_index < Nt) {
        size_t targets_left_to_process = Nt - processed_index;
        size_t blockSize = std::min(targets_left_to_process, max_block_size);
        float3* sub_target_ptr = targets + processed_index;
        size_t sub_target_size = blockSize;

        // std::cout << "in_sphere_mem_kernel " << numBlock << " " << blockSize
        // << std::endl;
        reach_mem_kernel<<<numBlock, blockSize>>>(
            body_centers, Nc, sub_target_ptr, sub_target_size, output.elements,
            dim, orientation);

        processed_index += blockSize;
    }
};

__global__ void find_min_kernel(Array<int>* arrays, int number_of_legs,
                                Array<int> output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < output.length; i += stride) {
        // for (int i = index; i < 1; i += stride) {
        int min_value = arrays[0].elements[i];

        for (int leg = 1; leg < number_of_legs; ++leg) {
            min_value = min(min_value, arrays[leg].elements[i]);
            // min_value = min_value + arrays[leg].elements[i];
        }

        output.elements[i] = min_value;
    }
}

#define CUDA_CHECK_ERROR(errorMessage)                                         \
    do {                                                                       \
        hipError_t err = hipGetLastError();                                  \
        if (err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error in %s: %s\n", errorMessage,            \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

// Array<int> robot_full_reachable(Array<float3> body_map,
//                                 Array<float3> target_map,
//                                 Array<LegDimensions> legs) {
//     {
//         float3* newpointer;
//         hipMalloc(&newpointer, body_map.length * sizeof(float3));
//         CUDA_CHECK_ERROR("hipMalloc body_map");
//         hipMemcpy(newpointer, body_map.elements,
//                    body_map.length * sizeof(float3), hipMemcpyHostToDevice);
//         CUDA_CHECK_ERROR("hipMemcpy body_map");
//         body_map.elements = newpointer;
//     }
//     {
//         float3* newpointer;
//         hipMalloc(&newpointer, target_map.length * sizeof(float3));
//         CUDA_CHECK_ERROR("hipMalloc target_map");
//         hipMemcpy(newpointer, target_map.elements,
//                    target_map.length * sizeof(float3),
//                    hipMemcpyHostToDevice);
//         CUDA_CHECK_ERROR("hipMemcpy target_map");
//         target_map.elements = newpointer;
//     }
//     CUDA_CHECK_ERROR("hipMalloc before leg");
//
//     Array<int>* res_bool_array;
//     res_bool_array = new Array<int>[legs.length];
//     int blockSize = 1024 / 1;
//     int numBlock =
//         (body_map.length * target_map.length + blockSize - 1) / blockSize;
//
//     for (int leg_num = 0; leg_num < legs.length; leg_num++) {
//         res_bool_array[leg_num].length = body_map.length;
//         hipMalloc(&(res_bool_array[leg_num].elements),
//                    sizeof(int) * res_bool_array[leg_num].length);
//         if (leg_num == 0) {
//             hipMemset(res_bool_array[leg_num].elements, 0,
//                        sizeof(int) * res_bool_array[leg_num].length);
//             float radius = legs.elements[0].body;
//             float plus_z = 120;
//             float minus_z = -60;
//             in_cylinder_accu_kernel<<<numBlock, blockSize>>>(
//                 body_map, target_map, res_bool_array[leg_num], radius,
//                 plus_z, minus_z);
//         } else {
//             hipMemcpy(res_bool_array[leg_num].elements,
//                        res_bool_array[0].elements,
//                        sizeof(int) * res_bool_array[leg_num].length,
//                        hipMemcpyDeviceToDevice);
//         }
//     }
//     hipDeviceSynchronize();
//     CUDA_CHECK_ERROR("Cylinder and alloc");
//     blockSize = 1024 / 2;
//     numBlock =
//         (body_map.length * target_map.length + blockSize - 1) / blockSize;
//     for (int leg_num = 0; leg_num < legs.length; leg_num++) {
//         CUDA_CHECK_ERROR("cudaKernel leg");
//         reachable_leg_kernel_accu<<<numBlock, blockSize>>>(
//             body_map, target_map, legs.elements[leg_num],
//             res_bool_array[leg_num]);
//     }
//     hipDeviceSynchronize();
//
//     CUDA_CHECK_ERROR("hipMalloc reachable kernel");
//
//     Array<int> final_count;
//     final_count.length = res_bool_array[0].length;
//
//     hipMalloc(&final_count.elements, sizeof(int) * final_count.length);
//     hipMemset(final_count.elements, 0, final_count.length * sizeof(int));
//     CUDA_CHECK_ERROR("hipMalloc final_count");
//
//     {
//         Array<int>* newpointer;
//         hipMalloc(&newpointer, legs.length * sizeof(Array<int>));
//         CUDA_CHECK_ERROR("hipMalloc legdim");
//         hipMemcpy(newpointer, res_bool_array, legs.length *
//         sizeof(Array<int>),
//                    hipMemcpyHostToDevice);
//         CUDA_CHECK_ERROR("hipMemcpy legdim");
//         res_bool_array = newpointer;
//     }
//
//     blockSize = 1024;
//     numBlock = (body_map.length + blockSize - 1) / blockSize;
//     find_min_kernel<<<numBlock, blockSize>>>(res_bool_array, legs.length,
//                                              final_count);
//     hipDeviceSynchronize();
//     CUDA_CHECK_ERROR("execution find_min_kernel");
//     int* newpointer = new int[final_count.length];
//     {
//         hipMemcpy(newpointer, final_count.elements,
//                    final_count.length * sizeof(int), hipMemcpyDeviceToHost);
//         CUDA_CHECK_ERROR("hipMemcpy final_count");
//         hipFree(final_count.elements);
//         final_count.elements = newpointer;
//     }
//
//     hipFree(body_map.elements);
//     hipFree(target_map.elements);
//     hipDeviceSynchronize();
//     CUDA_CHECK_ERROR("hipFree");
//     return final_count;
// }

class multi_rot_estimator {
  public:
    thrust::device_vector<float3> bodyWorking;
    thrust::device_vector<float3> targetWorking;
    thrust::device_vector<float3> targetRotated;
    thrust::device_vector<unsigned char>* resultLegArray;
    thrust::device_vector<unsigned char>& finalCount;

    thrust::device_vector<float3> bodyGlobal;
    thrust::device_vector<float3> targetGlobal;
    thrust::device_vector<float3>::iterator endBodyView;
    thrust::device_vector<float3>::iterator beginBodyView;
    Array<LegDimensions> legs;
    Array<LegDimensions> legsWorking;

    multi_rot_estimator(thrust::device_vector<float3> body,
                        thrust::device_vector<float3> target,
                        Array<LegDimensions> legsArray)
        : resultLegArray(new thrust::device_vector<unsigned char>[4]),
          finalCount(resultLegArray[0]) {

        bodyGlobal.resize(body.size());
        thrust::copy(body.begin(), body.end(), bodyGlobal.begin());

        targetWorking.resize(target.size());
        thrust::copy(target.begin(), target.end(), targetWorking.begin());

        targetGlobal.resize(target.size());
        thrust::copy(target.begin(), target.end(), targetGlobal.begin());

        legs = legsArray;
        legsWorking.length = legs.length;
        legsWorking.elements = new LegDimensions[legsWorking.length];
        for (int i = 0; i < legs.length; i++) {
            legsWorking.elements[i] = legs.elements[i];
        }

        bodyWorking.resize(bodyGlobal.size());
        thrust::copy(bodyGlobal.begin(), bodyGlobal.end(), bodyWorking.begin());

        targetRotated.resize(targetWorking.size());
        thrust::copy(targetWorking.begin(), targetWorking.end(),
                     targetRotated.begin());

        std::cout << "init: " << (bodyGlobal.end() - bodyGlobal.begin())
                  << std::endl;

        eliminateAlwaysColliding();
        eliminateFarBody();
        // resetWorkingData();
        eliminateFarTarget();
        resetWorkingData();
        // raise(1);
    }

    void resetWorkingData() {
        targetRotated.resize(targetWorking.end() - targetWorking.begin());
        thrust::copy(targetWorking.begin(), targetWorking.end(),
                     targetRotated.begin());
        // std::cout << "body W address " << &(bodyWorking) << std::endl;
        bodyWorking.resize(endBodyView - beginBodyView);
        thrust::copy(beginBodyView, endBodyView, bodyWorking.begin());
        for (int i = 0; i < legs.length; i++) {
            legsWorking.elements[i] = legs.elements[i];
        }
        return;
    }

    ~multi_rot_estimator() {
        delete[] resultLegArray;
        delete[] legsWorking.elements;
    }

    void flipWorkingSide() {
        beginBodyView = endBodyView;
        endBodyView = bodyGlobal.end();
    }

    void rotateData(Quaternion quat) {
        QuaternionFunctor my_func = QuaternionFunctor(quat);
        std::cout << quat.x << " | " << quat.y << " | " << quat.z << " | "
                  << quat.w << " | " << std::endl;

        thrust::transform(bodyWorking.begin(), bodyWorking.end(),
                          bodyWorking.begin(), my_func);
        thrust::transform(targetRotated.begin(), targetRotated.end(),
                          targetRotated.begin(), my_func);
        // std::cout << "Rotation done" << std::endl;
    };

    void eliminateAlwaysColliding() {
        thrust::device_vector<unsigned char> result(bodyGlobal.size());
        thrust::fill(result.begin(), result.end(), 0);
        float radius = 60;

        auto ptr = thrust::raw_pointer_cast(bodyGlobal.data());
        auto sizeBody = bodyGlobal.size();

        launch_optimized_mem_in_sphere(
            ptr, sizeBody, thrust::raw_pointer_cast(targetRotated.data()),
            targetRotated.size(), thrust::raw_pointer_cast(result.data()),
            radius);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("eliminateAlwaysColliding");

        auto newEnd = thrust::remove_if(
            bodyGlobal.begin(), bodyGlobal.end(), result.begin(),
            [] __device__(unsigned char x) { return x != 0; });
        bodyGlobal.erase(newEnd, bodyGlobal.end());
        newEnd = thrust::remove_if(
            bodyWorking.begin(), bodyWorking.end(), result.begin(),
            [] __device__(unsigned char x) { return x != 0; });
        bodyWorking.erase(newEnd, bodyWorking.end());
        beginBodyView = bodyGlobal.begin();
        endBodyView = bodyGlobal.end();

        std::cout << "eliminateAlwaysColliding: "
                  << (endBodyView - beginBodyView) << std::endl;
        // raise(1);
    }

    void eliminateFarBody() {
        // auto dim = legsWorking.elements[0];
        // float radius =
        // dim.body + dim.coxa_length + dim.femur_length + dim.tibia_length;
        float radius = 400;

        thrust::device_vector<unsigned char> not_far(bodyGlobal.size());
        thrust::fill(not_far.begin(), not_far.end(), 0);

        auto ptr_body = thrust::raw_pointer_cast(bodyGlobal.data());
        auto size_body = bodyGlobal.size();
        auto ptr_target = thrust::raw_pointer_cast(targetWorking.data());
        auto size_target = targetWorking.size();

        launch_optimized_mem_in_sphere( // 1 if close enough
            ptr_body, size_body, ptr_target, size_target,
            thrust::raw_pointer_cast(not_far.data()), radius);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("eliminateTooFar");

        auto newEnd = thrust::remove_if(
            bodyGlobal.begin(), bodyGlobal.end(), not_far.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        std::cout << "# radius: " << (radius) << std::endl;
        std::cout << "# body deleted: " << (bodyGlobal.end() - newEnd)
                  << std::endl;
        bodyGlobal.erase(newEnd, bodyGlobal.end());
        newEnd = thrust::remove_if(
            bodyWorking.begin(), bodyWorking.end(), not_far.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        bodyWorking.erase(newEnd, bodyWorking.end());
        beginBodyView = bodyGlobal.begin();
        endBodyView = bodyGlobal.end();
    }

    void eliminateFarTarget() {
        // auto dim = legsWorking.elements[0];
        // float radius =
        // dim.body + dim.coxa_length + dim.femur_length + dim.tibia_length;
        float radius = 400;

        thrust::device_vector<unsigned char> not_far(targetWorking.size());
        thrust::fill(not_far.begin(), not_far.end(), 0);

        auto ptr_body = thrust::raw_pointer_cast(bodyWorking.data());
        auto size_body = bodyWorking.size();
        auto ptr_target = thrust::raw_pointer_cast(targetWorking.data());
        auto size_target = targetWorking.size();

        launch_optimized_mem_in_sphere( // 1 if close enough
            ptr_target, size_target, ptr_body, size_body,
            thrust::raw_pointer_cast(not_far.data()), radius);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("eliminateTooFar");

        auto newEnd = thrust::remove_if(
            targetWorking.begin(), targetWorking.end(), not_far.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        std::cout << "# radius: " << (radius) << std::endl;
        std::cout << "# target deleted: " << (targetWorking.end() - newEnd)
                  << std::endl;
        targetWorking.erase(newEnd, targetWorking.end());
    }

    void eliminateFarAndColliding() {
        auto dim = legsWorking.elements[0];

        float s_coxa_pitch = sin(dim.coxa_pitch);
        float c_coxa_pitch = cos(dim.coxa_pitch);
        float radius_in = dim.body + c_coxa_pitch * dim.coxa_length +
                          dim.femur_length + dim.tibia_length;
        auto plus_with_abs_limit =
            dim.tibia_length * sin(dim.tibia_absolute_pos) +
            dim.femur_length * sin(std::min(pI / 2, dim.max_angle_femur));
        float plus_z_in = s_coxa_pitch * dim.coxa_length + plus_with_abs_limit;
        float minus_z_in = s_coxa_pitch * dim.coxa_length - dim.femur_length -
                           dim.tibia_length;

        float radius_out = dim.body;
        float plus_z_out = 250;
        float minus_z_out = -60;

        thrust::device_vector<unsigned char> result(bodyWorking.size());
        thrust::fill(result.begin(), result.end(), 0);

        auto ptr = thrust::raw_pointer_cast(bodyWorking.data());
        auto sizeBody = bodyWorking.size();

        auto func_inside = [radius_in, plus_z_in, minus_z_in] __device__(
                               float3 body, float3 target) -> bool {
            bool inside_inside_cyl =
                in_cylinder(radius_in, plus_z_in, minus_z_in, body, target);
            return inside_inside_cyl;
        };

        auto func_outside = [radius_out, plus_z_out, minus_z_out] __device__(
                                float3 body, float3 target) -> bool {
            bool outside_outside_cyl =
                in_cylinder(radius_out, plus_z_out, minus_z_out, body, target);
            return outside_outside_cyl;
        };

        launch_double_reduction(
            ptr, sizeBody, thrust::raw_pointer_cast(targetRotated.data()),
            targetRotated.size(), thrust::raw_pointer_cast(result.data()),
            func_inside, func_outside);

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("eliminateTooFar");

        endBodyView = thrust::partition(
            beginBodyView, endBodyView, result.begin(),
            [] __device__(unsigned char x) { return x == 1; });
        auto newEnd = thrust::remove_if(
            bodyWorking.begin(), bodyWorking.end(), result.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        std::cout << "eliminateFarAndColliding deleted: "
                  << (bodyWorking.end() - newEnd) << std::endl;
        bodyWorking.erase(newEnd, bodyWorking.end());
    }

    void eliminateTooFar() {
        auto dim = legsWorking.elements[0];
        float s_coxa_pitch = sin(dim.coxa_pitch);
        float c_coxa_pitch = cos(dim.coxa_pitch);
        float radius = dim.body + c_coxa_pitch * dim.coxa_length +
                       dim.femur_length + dim.tibia_length;
        auto plus_with_abs_limit =
            dim.tibia_length * sin(dim.tibia_absolute_pos) +
            dim.femur_length * sin(std::min(pI / 2, dim.max_angle_femur));
        float plus_z = s_coxa_pitch * dim.coxa_length + plus_with_abs_limit;
        float minus_z = s_coxa_pitch * dim.coxa_length - dim.femur_length -
                        dim.tibia_length;
        thrust::device_vector<unsigned char> not_far(bodyWorking.size());
        thrust::fill(not_far.begin(), not_far.end(), 0);

        auto ptr = thrust::raw_pointer_cast(bodyWorking.data());
        auto sizeBody = bodyWorking.size();

        auto func = [=] __device__(float3 body, float3 target) -> bool {
            return in_cylinder(radius, plus_z, minus_z, body, target);
        };

        launch_double_reduction(ptr, sizeBody,
                                thrust::raw_pointer_cast(targetRotated.data()),
                                targetRotated.size(),
                                thrust::raw_pointer_cast(not_far.data()), func);

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("eliminateTooFar");

        endBodyView = thrust::partition(
            beginBodyView, endBodyView, not_far.begin(),
            [] __device__(unsigned char x) { return x != 0; });
        auto newEnd = thrust::remove_if(
            bodyWorking.begin(), bodyWorking.end(), not_far.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        bodyWorking.erase(newEnd, bodyWorking.end());
        std::cout << "eliminateTooFar: " << (endBodyView - beginBodyView)
                  << std::endl;
    }

    void eliminateBodyColliding() {
        thrust::device_vector<unsigned char> result(bodyWorking.size());
        thrust::fill(result.begin(), result.end(), 0);
        float radius = legsWorking.elements[0].body;
        float plus_z = 250;
        float minus_z = -60;

        auto ptr = thrust::raw_pointer_cast(bodyWorking.data());
        auto sizeBody = bodyWorking.size();

        auto func = [=] __device__(float3 body, float3 target) -> bool {
            return in_cylinder(radius, plus_z, minus_z, body, target);
        };

        launch_double_reduction(ptr, sizeBody,
                                thrust::raw_pointer_cast(targetRotated.data()),
                                targetRotated.size(),
                                thrust::raw_pointer_cast(result.data()), func);

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("eliminateBodyColliding");

        endBodyView = thrust::partition(
            beginBodyView, endBodyView, result.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        auto newEnd = thrust::remove_if(
            bodyWorking.begin(), bodyWorking.end(), result.begin(),
            [] __device__(unsigned char x) { return x != 0; });
        bodyWorking.erase(newEnd, bodyWorking.end());

        std::cout << "eliminateBodyColliding: " << (endBodyView - beginBodyView)
                  << std::endl;
    }

    void computeIndividualLegReachability(size_t leg_num,
                                          Quaternion orientation) {
        thrust::device_vector<unsigned char>& this_leg_result =
            resultLegArray[leg_num];
        auto dim = legsWorking.elements[leg_num];

        auto ptr = thrust::raw_pointer_cast(bodyWorking.data());
        const size_t sizeBody = bodyWorking.size();

        Array<float3> b = {sizeBody, ptr};
        Array<float3> t = {targetRotated.size(),
                           thrust::raw_pointer_cast(targetRotated.data())};
        Array<unsigned char> r = {
            this_leg_result.size(),
            thrust::raw_pointer_cast(this_leg_result.data())};

        // auto func = [orientation, dim] __device__(float3 body,
        //                                           float3 target) -> bool {
        //     return reachable_rotate_leg(target, body, orientation, dim);
        // };
        // launch_double_reduction(
        //     ptr, sizeBody, thrust::raw_pointer_cast(targetRotated.data()),
        //     targetRotated.size(),
        //     thrust::raw_pointer_cast(this_leg_result.data()), func);

        // reachable_leg_kernel_accu<<<numBlock, blockSize>>>(
        //     b, t, legsWorking.elements[leg_num], r);
        launch_opti_mem_reach_kernel(b, t, dim, orientation, r);
        CUDA_CHECK_ERROR("computeIndividualLegReachability");
    }

    void resetLegsResult() {
        delete[] resultLegArray;
        resultLegArray =
            new thrust::device_vector<unsigned char>[legsWorking.length];
        for (int i = 0; i < legsWorking.length; ++i) {
            resultLegArray[i].resize(bodyWorking.size());
            thrust::fill(resultLegArray[i].begin(), resultLegArray[i].end(), 0);
        }
    }

    void computeAllLegReachability(Quaternion orientation) {
        for (int leg_num = 0; leg_num < legsWorking.length; leg_num++) {
            computeIndividualLegReachability(leg_num, orientation);
        }
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("computeAllLegReachability");
    }

    void agregateReachability() {
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(
                resultLegArray[0].begin(), resultLegArray[1].begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                resultLegArray[0].end(), resultLegArray[1].end())),
            resultLegArray[0].begin(), MinRowElement<unsigned char>());
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(
                resultLegArray[2].begin(), resultLegArray[3].begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                resultLegArray[2].end(), resultLegArray[3].end())),
            resultLegArray[2].begin(), MinRowElement<unsigned char>());
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(
                resultLegArray[0].begin(), resultLegArray[2].begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                resultLegArray[0].end(), resultLegArray[2].end())),
            resultLegArray[0].begin(), MinRowElement<unsigned char>());
    }
    void cleanAgregated() {
        endBodyView = thrust::partition(
            beginBodyView, endBodyView, finalCount.begin(),
            [] __device__(unsigned char x) { return x != 0; });
        auto newEnd = thrust::remove_if(
            bodyWorking.begin(), bodyWorking.end(), finalCount.begin(),
            [] __device__(unsigned char x) { return x == 0; });
        bodyWorking.erase(newEnd, bodyWorking.end());
    }
    void eliminateUnreachable(Quaternion orientation) {
        // auto start = std::chrono::high_resolution_clock::now();
        resetLegsResult();
        // auto end = std::chrono::high_resolution_clock::now();
        // auto duration =
        // std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        // std::cout << "reset took: " << duration.count()
        // << " milliseconds to finish." << std::endl;
        // start = std::chrono::high_resolution_clock::now();
        computeAllLegReachability(orientation);
        // end = std::chrono::high_resolution_clock::now();
        // duration =
        //     std::chrono::duration_cast<std::chrono::milliseconds>(end -
        //     start);
        // std::cout << "computeAllLegReachability took: " << duration.count()
        //           << " milliseconds to finish." << std::endl;
        // start = std::chrono::high_resolution_clock::now();
        agregateReachability();
        // end = std::chrono::high_resolution_clock::now();
        // duration =
        //     std::chrono::duration_cast<std::chrono::milliseconds>(end -
        //     start);
        // std::cout << "agregateReachability took: " << duration.count()
        //           << " milliseconds to finish." << std::endl;
        // start = std::chrono::high_resolution_clock::now();
        cleanAgregated();
        // end = std::chrono::high_resolution_clock::now();
        // duration =
        //     std::chrono::duration_cast<std::chrono::milliseconds>(end -
        //     start);
        // std::cout << "cleanAgregated took: " << duration.count()
        //           << " milliseconds to finish." << std::endl;
        // start = std::chrono::high_resolution_clock::now();
        std::cout << "final count: " << (endBodyView - beginBodyView)
                  << std::endl;
    }

    void rotateOneLegLimit(Quaternion quat, LegDimensions& leg) {
        Quaternion quatOfLegAzimut =
            quatFromVectAngle(make_float3(0, 0, 1), leg.body_angle);

        Quaternion result = qtMultiply(qtMultiply(quatOfLegAzimut, quat),
                                       qtInvert(quatOfLegAzimut));
        float3 rpy = rpyFromQuat(result);
        float pitch = rpy.y;

        leg.tibia_absolute_pos -= pitch;
        leg.tibia_absolute_neg -= pitch;
    }

    void rotateLegsLimits(Quaternion quat) {
        for (int i = 0; i < legsWorking.length; i++) {
            rotateOneLegLimit(quat, legsWorking.elements[i]);
        }
    }

    void runPipeline(Quaternion orientation) {
        CUDA_TIMING_INIT();
        CUDA_TIMING_START();
        resetWorkingData();
        CUDA_TIMING_STOP("\nresetWorkingData")
        CUDA_TIMING_START();
        rotateData(orientation);
        CUDA_TIMING_STOP("rotateData")
        CUDA_TIMING_START();
        rotateLegsLimits(orientation);
        CUDA_TIMING_STOP("rotateLegsLimits")
        CUDA_TIMING_START();
        eliminateFarAndColliding();
        // eliminateTooFar();
        // CUDA_TIMING_STOP("eliminateTooFar")
        // CUDA_TIMING_START();
        // eliminateBodyColliding();
        CUDA_TIMING_STOP("eliminateFarColliding")
        CUDA_TIMING_START();
        eliminateUnreachable(orientation);
        CUDA_TIMING_STOP("eliminateUnreachable")
        CUDA_TIMING_START();
        flipWorkingSide();
        CUDA_TIMING_STOP("flipWorkingSide")
        CUDA_TIMING_START();
    } // 1.427 in 132s

    thrust::host_vector<float3> getShavedResult() {
        thrust::host_vector<float3> oncpub(beginBodyView - bodyGlobal.begin());
        thrust::copy(bodyGlobal.begin(), beginBodyView, oncpub.begin());
        return oncpub;
    }
};

std::tuple<Array<float3>, Array<int>>
robot_full_struct(Array<float3> body_map, Array<float3> target_map,
                  Array<LegDimensions> legs) {

    thrust::host_vector<float3> Body(body_map.elements,
                                     body_map.elements + body_map.length);
    thrust::host_vector<float3> Target(target_map.elements,
                                       target_map.elements + target_map.length);
    thrust::device_vector<float3> Body_g = Body;
    thrust::device_vector<float3> Target_g = Target;

    multi_rot_estimator estimator = multi_rot_estimator(Body_g, Target_g, legs);
    Quaternion quatInit = quatFromVectAngle(make_float3(0, 0, 1), 0);
    // estimator.runPipeline(quatInit);

    float rollMin = -pI / 4;
    float rollMax = pI / 4;
    // float rollMin = 0;
    // float rollMax = 0;
    int rollSample = 4;

    float pitchMin = -pI / 4;
    float pitchMax = +pI / 4;
    // float pitchMin = 0;
    // float pitchMax = 0;
    int pitchSample = 4;

    float yawMin = 0;
    float yawMax = pI / 2;
    // float yawMax = 0;
    int yawSample = 4;

    for (int rollN = 0; rollN <= rollSample; rollN++) {
        float rollX = (float)rollN / (float)rollSample;
        float roll = rollMin + (rollMax - rollMin) * rollX;
        Quaternion quatRoll = quatFromVectAngle(make_float3(1, 0, 0), roll);
        quatRoll = qtMultiply(quatRoll, quatInit);

        for (int pitchN = 0; pitchN <= pitchSample; pitchN++) {
            float pitchX = (float)pitchN / (float)pitchSample;
            float pitch = pitchMin + (pitchMax - pitchMin) * pitchX;
            Quaternion quatPitch =
                quatFromVectAngle(make_float3(0, 1, 0), pitch);
            quatPitch = qtMultiply(quatPitch, quatRoll);

            for (int yawN = 0; yawN <= yawSample; yawN++) {
                float yawX = (float)yawN / (float)yawSample;
                float yaw = yawMin + (yawMax - yawMin) * yawX;
                std::cout << "" << std::endl;
                std::cout << "roll: " << roll << " | pitch: " << pitch
                          << " | yaw: " << yaw << std::endl;
                Quaternion quatYaw =
                    quatFromVectAngle(make_float3(0, 0, 1), yaw);
                quatYaw = qtMultiply(quatYaw, quatPitch);
                CUDA_TIMING_INIT();
                CUDA_TIMING_START();
                estimator.runPipeline(quatYaw);
                CUDA_TIMING_STOP("runPipeline");
            }
        }
    }
    // quat = quatFromVectAngle(make_float3(0, 0, 1), 1 * pI / 8);
    // estimator.runPipeline(quat);
    // quat = quatFromVectAngle(make_float3(0, 0, 1), 2 * pI / 8);
    // estimator.runPipeline(quat);
    // quat = quatFromVectAngle(make_float3(0, 0, 1), 3 * pI / 8);
    // estimator.runPipeline(quat);

    // estimator.flipWorkingSide();
    thrust::host_vector<float3> outBody = estimator.getShavedResult();
    thrust::host_vector<int> outCount(outBody.size());
    thrust::fill(outCount.begin(), outCount.end(), 3);

    Array<float3> out_body = thustVectToArray(outBody);
    Array<int> out_count = thustVectToArray(outCount);
    auto out = std::make_tuple(out_body, out_count);
    std::cout << "cuda done" << std::endl;
    return out;
}
//
// std::tuple<Array<float3>, Array<int>>
// robot_full_cccl(Array<float3> body_map, Array<float3> target_map,
//                 Array<LegDimensions> legs) {
//     thrust::host_vector<float3> Body(body_map.elements,
//                                      body_map.elements + body_map.length);
//     thrust::host_vector<float3> Target(target_map.elements,
//                                        target_map.elements +
//                                        target_map.length);
//
//     thrust::device_vector<float3> Body_g = Body;
//     thrust::device_vector<float3> Target_g = Target;
//
//     std::cout << Body_g.size() << std::endl;
//
//     thrust::device_vector<float3>::iterator newEndBody = Body_g.end();
//     thrust::device_vector<float3>::iterator newBeginBody = Body_g.begin();
//     // newBeginBody += (newEndBody - newBeginBody) / 2;
//
//     // Quaternion quat = quatFromVectAngle(make_float3(0, 0, 1), pI/2);
//     Quaternion quat = quatFromVectAngle(make_float3(0, 0, 1), 0.0);
//     auto my_func = QuaternionFunctor(quat);
//     std::cout << quat.x << " | " << quat.y << " | " << quat.z << " | " <<
//     quat.w
//               << " | " << std::endl;
//
//     thrust::transform(newBeginBody, newEndBody, newBeginBody, my_func);
//     thrust::transform(Target_g.begin(), Target_g.end(), Target_g.begin(),
//                       my_func);
//
//     thrust::device_vector<int>* result_ar;
//     result_ar = new thrust::device_vector<int>[legs.length];
//     size_t blockSize = 1024 / 1;
//     size_t numBlock =
//         ((newEndBody - newBeginBody) * Target_g.size() + blockSize - 1) /
//         blockSize;
//     thrust::device_vector<int>::iterator newEndResult;
//     // int numBlock = (Body_g.size() + blockSize - 1) / blockSize;
//     {
//         auto dim = legs.elements[0];
//         float sinr = sin(dim.coxa_pitch);
//         float cosr = cos(dim.coxa_pitch);
//         float radius = dim.body + cosr * dim.coxa_length + dim.femur_length +
//                        dim.tibia_length;
//         float plus_z =
//             sinr * dim.coxa_length + dim.femur_length + dim.tibia_length;
//         float minus_z =
//             sinr * dim.coxa_length - dim.femur_length - dim.tibia_length;
//         thrust::device_vector<int> not_far(newEndBody - newBeginBody);
//         thrust::fill(not_far.begin(), not_far.end(), 0);
//         hipDeviceSynchronize();
//
//         // in_cylinder_rec<<<1, 1>>>( // -1 if close enough
//         auto ptr = thrust::raw_pointer_cast(Body_g.data()) +
//                    (newBeginBody - Body_g.begin());
//         auto s = newEndBody - newBeginBody;
//         in_cylinder_cccl_kernel<<<numBlock, blockSize>>>( // -1 if close
//         enough
//             ptr, s, thrust::raw_pointer_cast(Target_g.data()),
//             Target_g.size(), thrust::raw_pointer_cast(not_far.data()),
//             radius, plus_z, minus_z);
//         hipDeviceSynchronize();
//
//         newEndBody =
//             thrust::partition(newBeginBody, newEndBody, not_far.begin(),
//                               [] __device__(int x) { return x != 0; });
//         // Body_g.erase(newEndBody, Body_g.end());
//         std::cout << "Part 1 " << (newEndBody - newBeginBody) << std::endl;
//     }
//     CUDA_CHECK_ERROR("first cylinder and alloc");
//
//     for (int leg_num = 0; leg_num < legs.length; leg_num++) {
//         thrust::device_vector<int>& this_leg_result = result_ar[leg_num];
//         this_leg_result.resize(newEndBody - newBeginBody);
//         if (leg_num == 0) {
//             thrust::fill(this_leg_result.begin(), this_leg_result.end(), 0);
//             float radius = legs.elements[0].body;
//             float plus_z = 120;
//             float minus_z = -60;
//             auto ptr = thrust::raw_pointer_cast(Body_g.data()) +
//                        (newBeginBody - Body_g.begin());
//             auto s = newEndBody - newBeginBody;
//             in_cylinder_cccl_kernel<<<numBlock, blockSize>>>(
//                 ptr, s, thrust::raw_pointer_cast(Target_g.data()),
//                 Target_g.size(),
//                 thrust::raw_pointer_cast(this_leg_result.data()), radius,
//                 plus_z, minus_z);
//             hipDeviceSynchronize();
//
//             newEndBody = thrust::partition(
//                 newBeginBody, newEndBody, this_leg_result.begin(),
//                 [] __device__(int x) { return x != -1; });
//             // Body_g.erase(newEndBody, Body_g.end());
//             std::cout << "Part 2 " << (newEndBody - newBeginBody) <<
//             std::endl; newEndResult = thrust::remove(this_leg_result.begin(),
//                                           this_leg_result.end(), -1);
//             this_leg_result.erase(newEndResult, this_leg_result.end());
//
//         } else {
//             thrust::copy(result_ar[0].begin(), result_ar[0].end(),
//                          this_leg_result.begin());
//         }
//     }
//     CUDA_CHECK_ERROR("Cylinder and alloc");
//     blockSize = 1024 / 2;
//     numBlock = ((newEndBody - newBeginBody) * Target_g.size() + blockSize -
//     1) /
//                blockSize;
//
//     for (int leg_num = 0; leg_num < legs.length; leg_num++) {
//         CUDA_CHECK_ERROR("cudaKernel leg");
//         auto ptr = thrust::raw_pointer_cast(Body_g.data()) +
//                    (newBeginBody - Body_g.begin());
//         const size_t s = (newEndBody - newBeginBody);
//         Array<float3> b = {s, ptr};
//         Array<float3> t = {Target_g.size(),
//                            thrust::raw_pointer_cast(Target_g.data())};
//         Array<int> r = {result_ar[leg_num].size(),
//                         thrust::raw_pointer_cast(result_ar[leg_num].data())};
//         reachable_leg_kernel_accu<<<numBlock, blockSize>>>(
//             b, t, legs.elements[leg_num], r);
//     }
//     hipDeviceSynchronize();
//
//     CUDA_CHECK_ERROR("hipMalloc final_count");
//
//     thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(
//                           result_ar[0].begin(), result_ar[1].begin())),
//                       thrust::make_zip_iterator(thrust::make_tuple(
//                           result_ar[0].end(), result_ar[1].end())),
//                       result_ar[0].begin(), MinRowElement<int>());
//     thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(
//                           result_ar[2].begin(), result_ar[3].begin())),
//                       thrust::make_zip_iterator(thrust::make_tuple(
//                           result_ar[2].end(), result_ar[3].end())),
//                       result_ar[2].begin(), MinRowElement<int>());
//     thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(
//                           result_ar[0].begin(), result_ar[2].begin())),
//                       thrust::make_zip_iterator(thrust::make_tuple(
//                           result_ar[0].end(), result_ar[2].end())),
//                       result_ar[0].begin(), MinRowElement<int>());
//     thrust::device_vector<int> final_count = result_ar[0];
//     // blockSize = 1024;
//     // numBlock = ((newEndBody - newBeginBody) + blockSize - 1) / blockSize;
//     // find_min_kernel<<<numBlock, blockSize>>>(result_ar_gpu, legs.length,
//     //                                          final_count);
//     hipDeviceSynchronize();
//     CUDA_CHECK_ERROR("execution find_min_kernel");
//
//     newEndBody =
//         thrust::partition(newBeginBody, newEndBody, final_count.begin(),
//                           [] __device__(int x) { return x != 0; });
//     // Body_g.erase(newEndBody, Body_g.end());
//     std::cout << (newEndBody - newBeginBody) << std::endl;
//     newEndResult = thrust::remove(final_count.begin(), final_count.end(), 0);
//     final_count.erase(newEndResult, final_count.end());
//
//     auto my_UnFunc = UnQuaternionFunctor(quat);
//     thrust::transform(newBeginBody, newEndBody, newBeginBody, my_UnFunc);
//     thrust::transform(Target_g.begin(), Target_g.end(), Target_g.begin(),
//                       my_UnFunc);
//
//     thrust::host_vector<float3> oncpub(newEndBody - newBeginBody);
//     thrust::copy(newBeginBody, newEndBody, oncpub.begin());
//     Array<float3> out_body = thustVectToArray(oncpub);
//
//     thrust::host_vector<int> oncpuc = final_count;
//     Array<int> out_count = thustVectToArray(oncpuc);
//
//     CUDA_CHECK_ERROR("hipFree");
//     auto out = std::make_tuple(out_body, out_count);
//     std::cout << "cuda done" << std::endl;
//     return out;
// }
