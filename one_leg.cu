#include "hip/hip_runtime.h"
#include "HeaderCPP.h"
#include "HeaderCUDA.h"
#include "one_leg.cu.h"

__device__ void place_over_coxa(float3& coordinates, const LegDimensions& dim) {
    // Coxa as the frame of reference without rotation
    coordinates.x -= dim.body;
    float sin_coxa_memory;
    float cosine_coxa_memory;
    sincosf(-dim.coxa_pitch, &sin_coxa_memory, &cosine_coxa_memory);
    float buffer = coordinates.x * sin_coxa_memory;
    coordinates.x =
        coordinates.x * cosine_coxa_memory - coordinates.z * sin_coxa_memory;
    coordinates.z = buffer + coordinates.z * cosine_coxa_memory;
}

__device__ float find_coxa_angle(const float3& coordinates) {
    // finding coxa angle
    return atan2f(coordinates.y, coordinates.x);
}

__device__ void clamp_on_circle(const float* center, const float& radius,
                                const bool& clamp_direction, float& x,
                                float& y) {
    x -= center[0];
    y -= center[1];
    float magnitude = sqrtf(x * x + y * y);
    bool radius_direction = !signbit(radius - magnitude);

    if (clamp_direction != radius_direction or magnitude < 0.001f) {
        x = 0;
        y = 0;
        return;
    }

    x -= radius * x / magnitude;
    y -= radius * y / magnitude;
}

__device__ void place_on_vert_plane(float& x, float& z,
                                    const LegDimensions& dim) {
    // Femur as the frame of reference witout rotation
    x -= dim.coxa_length;

    float femur_angle_raw = atan2f(z, x);
    float femur_distance = norm3df(z, x, 0);

    bool positiv_side = !signbit(z);
    const float* saturated_femur_of_interest =
        (positiv_side) ? dim.positiv_saturated_femur
                       : dim.negativ_saturated_femur;
    const float& tibia_saturation =
        (positiv_side) ? dim.max_angle_tibia : -dim.min_angle_tibia;
    const float& femur_saturation =
        (positiv_side) ? dim.max_angle_femur : -dim.min_angle_femur;

    float tibia_angle_raw = atan2f(z - saturated_femur_of_interest[1],
                                   x - saturated_femur_of_interest[0]);
    if (!positiv_side) {
        tibia_angle_raw *= -1;
        femur_angle_raw *= -1;
    }

    bool too_close = femur_distance < dim.min_femur_to_gripper_dist;

    bool femur_condition = (femur_angle_raw <= femur_saturation);
    bool femur_cond_wider =
        femur_angle_raw <= (femur_saturation + dim.femur_overmargin_negative);
    bool tibia_condition =
        (((tibia_angle_raw <= (tibia_saturation + femur_saturation)) and
          !signbit(tibia_angle_raw)) or
         ((tibia_angle_raw <=
           (tibia_saturation + femur_saturation - 2 * pIgpu))));

    float center[2] = {0, 0};
    float radius = dim.min_femur_to_gripper_dist;
    bool& clamp_direction = too_close;

    if (femur_condition and not too_close) {
        radius = dim.max_femur_to_gripper_dist;
    } else if (tibia_condition and not too_close) {
        center[0] = saturated_femur_of_interest[0];
        center[1] = saturated_femur_of_interest[1];
        radius = dim.tibia_length;
    } else if (not femur_cond_wider and not tibia_condition) {
        center[0] = cosf(dim.femur_overmargin_negative + femur_saturation) *
                    dim.min_femur_to_gripper_dist;
        center[1] = sinf(dim.femur_overmargin_negative + femur_saturation) *
                    dim.min_femur_to_gripper_dist;
        center[1] = copysignf(center[1], z);
        radius = 0;
        clamp_direction = false;
    };

    clamp_on_circle(center, radius, clamp_direction, x, z);
}

// __device__ void place_on_vert_plane_old(float& x, float& z,
//                                         const LegDimensions& dim) {
//     // Femur as the frame of reference witout rotation
//     x -= dim.coxa_length;
//
//     // finding femur angle
//     float required_overangle_femur = atan2f(z, x);
//     float angle_overshoot = required_overangle_femur;
//
//     float overmargin = dim.femur_overmargin;
//
//     // saturating femur angle for dist
//     required_overangle_femur =
//         fmaxf(fminf(required_overangle_femur,
//                     dim.max_angle_femur_w_margin + overmargin),
//               dim.min_angle_femur_w_margin - overmargin);
//
//     angle_overshoot =
//         abs(angle_overshoot -
//             fmaxf(fminf(required_overangle_femur, dim.max_angle_femur),
//                   dim.min_angle_femur));
//
//     // canceling femur rotation for dist
//     float cos_angle_fem;
//     float sin_angle_fem;
//     sincosf(required_overangle_femur, &sin_angle_fem, &cos_angle_fem);
//
//     // How close we are from the overmargin. 0 means we're at the saturated
//     // femur, 1 means we're at the saturated femur and tibia
//     float ratio = fmin(fmax(angle_overshoot / overmargin, 0.f), 1.f);
//     float radius_adjust_factor = dim.middle_TG_radius * ratio * sinf(ratio);
//
//     // middle_TG is now the frame of reference
//     // if the middle radius is reduced by x in the overgin, middleTG is
//     brought
//     // back by x in order to keep the inner circle
//     (min_femur_to_gripper_dist)
//     // unchanged
//     x -= (dim.middle_TG - radius_adjust_factor) * cos_angle_fem;
//     z -= (dim.middle_TG - radius_adjust_factor) * sin_angle_fem;
//
//     // inside this radius the distance is zero
//     float zeroing_radius =
//         fmax(dim.middle_TG_radius_w_margin - radius_adjust_factor, 0.f);
//     float magnitude = fmax(norm3df(x, z, 0.f), zeroing_radius);
//     // the part of the vector inside the radius gets substracted
//     x -= zeroing_radius * x / magnitude;
//     z -= zeroing_radius * z / magnitude;
// }

__device__ void cancel_coxa_rotation(float3& coordinates,
                                     const float& coxa_angle,
                                     float& cosine_coxa_memory,
                                     float& sin_coxa_memory) {
    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    sincosf(-coxa_angle, &sin_coxa_memory, &cosine_coxa_memory);
    float buffer = coordinates.x * sin_coxa_memory;
    coordinates.x =
        coordinates.x * cosine_coxa_memory - coordinates.y * sin_coxa_memory;
    coordinates.y = buffer + coordinates.y * cosine_coxa_memory;
}

__device__ void restore_coxa_rotation(float3& coordinates,
                                      float& cosine_coxa_memory,
                                      float& sin_coxa_memory) {

    float buffer = coordinates.y * sin_coxa_memory;
    coordinates.y =
        -coordinates.x * sin_coxa_memory + coordinates.y * cosine_coxa_memory;
    coordinates.x = coordinates.x * cosine_coxa_memory + buffer;
}

__device__ void finish_finding_closest(float3& coordinates,
                                       const LegDimensions& dim,
                                       const float& coxa_angle) {
    // saturating coxa angle for dist
    float saturated_coxa_angle =
        fmaxf(fminf(coxa_angle, dim.max_angle_coxa_w_margin),
              dim.min_angle_coxa_w_margin);

    float cosine_coxa_memory;
    float sin_coxa_memory;
    cancel_coxa_rotation(coordinates, saturated_coxa_angle, cosine_coxa_memory,
                         sin_coxa_memory);

    place_on_vert_plane(coordinates.x, coordinates.z, dim);

    restore_coxa_rotation(coordinates, cosine_coxa_memory, sin_coxa_memory);
}

__device__ float3 dist_double_solf3(const float3& point,
                                    const LegDimensions& dim) {
    float3 closest = point;
    place_over_coxa(closest, dim);
    float3 closest_flip = closest;

    float coxangle = find_coxa_angle(closest);
    float coxangle_flip = (coxangle > 0) ? coxangle - pIgpu : coxangle + pIgpu;

    finish_finding_closest(closest, dim, coxangle);
    finish_finding_closest(closest_flip, dim, coxangle_flip);

    float3* result_to_use =
        (norm3df(closest.x, closest.y, closest.z) <
         norm3df(closest_flip.x, closest_flip.y, closest_flip.z))
            ? &closest
            : &closest_flip;

    return *result_to_use;
}

__device__ bool reachability_vect(const float3& point, const LegDimensions& dim)
// angle flipping on negative x values
{
    // Coxa as the frame of reference without rotation
    float3 result;
    result = point;
    place_over_coxa(result, dim);
    bool flip_flag = -signbit(result.x);

    if (flip_flag) {
        result.x *= -1;
        result.y *= -1;
    }

    // finding coxa angle
    float required_angle_coxa = find_coxa_angle(result);

    if (flip_flag) {
        result.x *= -1;
        result.y *= -1;
    }

    // flipping angle if above +-90deg

    if ((required_angle_coxa > dim.max_angle_coxa) ||
        (required_angle_coxa < dim.min_angle_coxa)) {
        return false;
    }

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox;
    float sin_angle_cox;
    cancel_coxa_rotation(result, required_angle_coxa, cos_angle_cox,
                         sin_angle_cox);

    // Femur as the frame of reference witout rotation
    result.x -= dim.coxa_length;

    float linnorm = norm3df(result.x, result.y, result.z);

    if ((linnorm < dim.min_femur_to_gripper_dist) ||
        (linnorm > dim.max_femur_to_gripper_dist)) {
        return false;
    }

    // finding femur angle
    float required_angle_femur = atan2f(result.z, result.x);

    if ((required_angle_femur >= dim.min_angle_femur) &&
        (required_angle_femur <= dim.max_angle_femur)) {
        return true;
    }

    // distance to femur at the most extrem position, this value is pre_computed
    linnorm = fminf(norm3df(result.x - dim.positiv_saturated_femur[0], 0,
                            result.z - dim.positiv_saturated_femur[1]),
                    norm3df(result.x - dim.negativ_saturated_femur[0], 0,
                            result.z - dim.negativ_saturated_femur[1]));

    return linnorm <= dim.tibia_length;
}

__device__ bool reachability_absolute_tibia_limit(const float3& point,
                                                  const LegDimensions& dim)
// the tibia cannot exceed a specified cangle relative to the BODY
// this ensures that the tibia is for example always pointing down
{
    // Coxa as the frame of reference without rotation
    float3 result;
    result = point;
    place_over_coxa(result, dim);
    bool flip_flag = -signbit(result.x);

    if (flip_flag) {
        result.x *= -1;
        result.y *= -1;
    }

    // finding coxa angle
    float required_angle_coxa = find_coxa_angle(result);

    if (flip_flag) {
        result.x *= -1;
        result.y *= -1;
    }

    if ((required_angle_coxa > dim.max_angle_coxa) ||
        (required_angle_coxa < dim.min_angle_coxa)) {
        return false;
    }

    // canceling coxa rotation for dist
    // Coxa as the frame of reference with rotation
    float cos_angle_cox;
    float sin_angle_cox;
    cancel_coxa_rotation(result, required_angle_coxa, cos_angle_cox,
                         sin_angle_cox);

    // Femur as the frame of reference witout rotation
    result.x -= dim.coxa_length;

    float linnorm = norm3df(result.x, result.y, result.z);

    if ((linnorm < dim.min_femur_to_gripper_dist) ||
        (linnorm > dim.max_femur_to_gripper_dist)) {
        return false;
    }

    float deported_center_negativ[2];
    sincosf(dim.tibia_absolute_neg, &sin_angle_cox, &cos_angle_cox);
    deported_center_negativ[0] = dim.tibia_length * cos_angle_cox;
    deported_center_negativ[1] = dim.tibia_length * sin_angle_cox;
    // should not be inside
    bool in_negativ_tib_circle =
        norm3df(result.x - deported_center_negativ[0], 0,
                result.z - deported_center_negativ[1]) <= dim.femur_length;

    // if (in_negativ_tib_circle) {return false;}

    // finding femur angle
    float required_angle_femur = atan2f(result.z, result.x);
    bool inside_femur = (required_angle_femur >= dim.min_angle_femur) &&
                        (required_angle_femur <= dim.tibia_absolute_pos);
    // if (inside_femur) {return true;}

    // - saturation is alowed
    bool in_negative_sat_circle =
        norm3df(result.x - dim.negativ_saturated_femur[0], 0,
                result.z - dim.negativ_saturated_femur[1]) <= dim.tibia_length;
    // if (in_negative_sat_circle) {return true;}

    // distance to femur at the most extrem  femur position should be less than
    // tibia length to be in the saturation circle, circle center is
    // pre_computed

    // We don´t want + saturation
    bool in_positive_sat_circle =
        norm3df(result.x - dim.positiv_saturated_femur[0], 0,
                result.z - dim.positiv_saturated_femur[1]) <= dim.tibia_length;

    float deported_center_positiv[2];
    sincosf(dim.tibia_absolute_pos, &sin_angle_cox, &cos_angle_cox);
    deported_center_positiv[0] = dim.tibia_length * cos_angle_cox;
    deported_center_positiv[1] = dim.tibia_length * sin_angle_cox;
    // need it inside
    bool in_positiv_tib_circle =
        norm3df(result.x - deported_center_positiv[0], 0,
                result.z - deported_center_positiv[1]) <= dim.femur_length;

    bool reachability =
        (not in_negativ_tib_circle and not in_positive_sat_circle) and
        ((in_negative_sat_circle or inside_femur or in_positiv_tib_circle));
    return reachability;
}

__global__ void dist_kernel(const Array<float3> input,
                            const LegDimensions dimensions,
                            Array<float3> const output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] = dist_double_solf3(input.elements[i], dimensions);
        // output.elements[i] = input.elements[i];
    }
}

__global__ void reachability_kernel(const Array<float3> input,
                                    const LegDimensions dimensions,
                                    Array<bool> const output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] = reachability_vect(input.elements[i], dimensions);
    }
}

__global__ void reachability_abs_tib_kernel(const Array<float3> input,
                                            const LegDimensions dimensions,
                                            Array<bool> const output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < input.length; i += stride) {
        output.elements[i] =
            reachability_absolute_tibia_limit(input.elements[i], dimensions);
    }
}

__device__ float3 forward_kinematics(const float coxa, const float femur,
                                     const float tibia,
                                     const LegDimensions dim) {
    float3 result{0, 0, 0};
    result.x += dim.body;
    float cos_horiz, sin_horiz;
    sincosf(coxa, &sin_horiz, &cos_horiz);
    result.x += cos_horiz * dim.coxa_length;
    result.y += sin_horiz * dim.coxa_length;

    float cos, sin;
    sincosf(femur, &sin, &cos);
    float horiz_distance = cos * dim.femur_length;
    float vert_distance = sin * dim.femur_length;
    result.x += cos_horiz * horiz_distance;
    result.y += sin_horiz * horiz_distance;
    result.z += vert_distance;

    sincosf(tibia + femur, &sin, &cos);
    horiz_distance = cos * dim.tibia_length;
    vert_distance = sin * dim.tibia_length;
    result.x += cos_horiz * horiz_distance;
    result.y += sin_horiz * horiz_distance;
    result.z += vert_distance;

    return result;
};

__global__ void forward_kine_kernel(const Array<float3> angles_3_input,
                                    const LegDimensions dim,
                                    Array<float3> const output_xyz) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < angles_3_input.length; i += stride) {
        output_xyz.elements[i] = forward_kinematics(
            angles_3_input.elements[i].x, angles_3_input.elements[i].y,
            angles_3_input.elements[i].z, dim);
    }
}
